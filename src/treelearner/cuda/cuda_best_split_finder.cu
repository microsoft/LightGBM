
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include <LightGBM/cuda/cuda_algorithms.hpp>
#include "cuda_best_split_finder.hpp"

namespace LightGBM {

__device__ void ReduceBestGainWarp(double gain, bool found, uint32_t thread_index, double* out_gain, bool* out_found, uint32_t* out_thread_index) {
  const uint32_t mask = 0xffffffff;
  const uint32_t warpLane = threadIdx.x % warpSize;
  for (uint32_t offset = warpSize / 2; offset > 0; offset >>= 1) {
    const bool other_found = __shfl_down_sync(mask, found, offset);
    const double other_gain = __shfl_down_sync(mask, gain, offset);
    const uint32_t other_thread_index = __shfl_down_sync(mask, thread_index, offset);
    if ((other_found && found && other_gain > gain) || (!found && other_found)) {
      found = other_found;
      gain = other_gain;
      thread_index = other_thread_index;
    }
  }
  if (warpLane == 0) {
    *out_gain = gain;
    *out_found = found;
    *out_thread_index = thread_index;
  }
}

__device__ uint32_t ReduceBestGainBlock(double gain, bool found, uint32_t thread_index) {
  const uint32_t mask = 0xffffffff;
  for (uint32_t offset = warpSize / 2; offset > 0; offset >>= 1) {
    const bool other_found = __shfl_down_sync(mask, found, offset);
    const double other_gain = __shfl_down_sync(mask, gain, offset);
    const uint32_t other_thread_index = __shfl_down_sync(mask, thread_index, offset);
    if ((other_found && found && other_gain > gain) || (!found && other_found)) {
      found = other_found;
      gain = other_gain;
      thread_index = other_thread_index;
    }
  }
  return thread_index;
}

__device__ uint32_t ReduceBestGain(double gain, bool found, uint32_t thread_index,
    double* shared_gain_buffer, bool* shared_found_buffer, uint32_t* shared_thread_index_buffer) {
  const uint32_t warpID = threadIdx.x / warpSize;
  const uint32_t warpLane = threadIdx.x % warpSize;
  const uint32_t num_warp = blockDim.x / warpSize;
  ReduceBestGainWarp(gain, found, thread_index, shared_gain_buffer + warpID, shared_found_buffer + warpID, shared_thread_index_buffer + warpID);
  __syncthreads();
  if (warpID == 0) {
    gain = warpLane < num_warp ? shared_gain_buffer[warpLane] : kMinScore;
    found = warpLane < num_warp ? shared_found_buffer[warpLane] : false;
    thread_index = warpLane < num_warp ? shared_thread_index_buffer[warpLane] : 0;
    thread_index = ReduceBestGainBlock(gain, found, thread_index);
  }
  return thread_index;
}

__device__ void ReduceBestGainForLeaves(double* gain, int* leaves, int cuda_cur_num_leaves) {
  const unsigned int tid = threadIdx.x;
  for (unsigned int s = 1; s < cuda_cur_num_leaves; s *= 2) {
    if (tid % (2 * s) == 0 && (tid + s) < cuda_cur_num_leaves) {
      const uint32_t tid_s = tid + s;
      if ((leaves[tid] == -1 && leaves[tid_s] != -1) || (leaves[tid] != -1 && leaves[tid_s] != -1 && gain[tid_s] > gain[tid])) {
        gain[tid] = gain[tid_s];
        leaves[tid] = leaves[tid_s];
      }
    }
    __syncthreads();
  }
}

__device__ void ReduceBestGainForLeavesWarp(double gain, int leaf_index, double* out_gain, int* out_leaf_index) {
  const uint32_t mask = 0xffffffff;
  const uint32_t warpLane = threadIdx.x % warpSize;
  for (uint32_t offset = warpSize / 2; offset > 0; offset >>= 1) {
    const int other_leaf_index = __shfl_down_sync(mask, leaf_index, offset);
    const double other_gain = __shfl_down_sync(mask, gain, offset);
    if ((leaf_index != -1 && other_leaf_index != -1 && other_gain > gain) || (leaf_index == -1 && other_leaf_index != -1)) {
      gain = other_gain;
      leaf_index = other_leaf_index;
    }
  }
  if (warpLane == 0) {
    *out_gain = gain;
    *out_leaf_index = leaf_index;
  }
}

__device__ int ReduceBestGainForLeavesBlock(double gain, int leaf_index) {
  const uint32_t mask = 0xffffffff;
  for (uint32_t offset = warpSize / 2; offset > 0; offset >>= 1) {
    const int other_leaf_index = __shfl_down_sync(mask, leaf_index, offset);
    const double other_gain = __shfl_down_sync(mask, gain, offset);
    if ((leaf_index != -1 && other_leaf_index != -1 && other_gain > gain) || (leaf_index == -1 && other_leaf_index != -1)) {
      gain = other_gain;
      leaf_index = other_leaf_index;
    }
  }
  return leaf_index;
}

__device__ int ReduceBestGainForLeaves(double gain, int leaf_index, double* shared_gain_buffer, int* shared_leaf_index_buffer) {
  const uint32_t warpID = threadIdx.x / warpSize;
  const uint32_t warpLane = threadIdx.x % warpSize;
  const uint32_t num_warp = blockDim.x / warpSize;
  ReduceBestGainForLeavesWarp(gain, leaf_index, shared_gain_buffer + warpID, shared_leaf_index_buffer + warpID);
  __syncthreads();
  if (warpID == 0) {
    gain = warpLane < num_warp ? shared_gain_buffer[warpLane] : kMinScore;
    leaf_index = warpLane < num_warp ? shared_leaf_index_buffer[warpLane] : -1;
    leaf_index = ReduceBestGainForLeavesBlock(gain, leaf_index);
  }
  return leaf_index;
}

__device__ double ThresholdL1(double s, double l1) {
  const double reg_s = fmax(0.0, fabs(s) - l1);
  if (s >= 0.0f) {
    return reg_s;
  } else {
    return -reg_s;
  }
}

__device__ double CUDABestSplitFinder::CalculateSplittedLeafOutput(double sum_gradients,
                                          double sum_hessians, double l1, const bool use_l1,
                                          double l2) {
  double ret;
  if (use_l1) {
    ret = -ThresholdL1(sum_gradients, l1) / (sum_hessians + l2);
  } else {
    ret = -sum_gradients / (sum_hessians + l2);
  }
  return ret;
}

__device__ double GetLeafGainGivenOutput(double sum_gradients,
                                      double sum_hessians, double l1, const bool use_l1,
                                      double l2, double output) {
  if (use_l1) {
    const double sg_l1 = ThresholdL1(sum_gradients, l1);
    return -(2.0 * sg_l1 * output + (sum_hessians + l2) * output * output);
  } else {
    return -(2.0 * sum_gradients * output +
              (sum_hessians + l2) * output * output);
  }
}

__device__ double GetLeafGain(double sum_gradients, double sum_hessians,
                          double l1, const bool use_l1, double l2) {
  if (use_l1) {
    const double sg_l1 = ThresholdL1(sum_gradients, l1);
    return (sg_l1 * sg_l1) / (sum_hessians + l2);
  } else {
    return (sum_gradients * sum_gradients) / (sum_hessians + l2);
  }
}

__device__ double GetSplitGains(double sum_left_gradients,
                            double sum_left_hessians,
                            double sum_right_gradients,
                            double sum_right_hessians,
                            double l1, const bool use_l1, double l2) {
  return GetLeafGain(sum_left_gradients,
                     sum_left_hessians,
                     l1, use_l1, l2) +
         GetLeafGain(sum_right_gradients,
                     sum_right_hessians,
                     l1, use_l1, l2);
}

__device__ void FindBestSplitsForLeafKernelInner(
  // input feature information
  const hist_t* feature_hist_ptr,
  const uint32_t feature_num_bin,
  const uint8_t feature_mfb_offset,
  const uint32_t feature_default_bin,
  const int inner_feature_index,
  // input config parameter values
  const double lambda_l1,
  const double lambda_l2,
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  // input parent node information
  const double parent_gain,
  const double sum_gradients,
  const double sum_hessians,
  const data_size_t num_data,
  // input task information
  const bool reverse,
  const bool skip_default_bin,
  const bool na_as_missing,
  const uint8_t assume_out_default_left,
  // output parameters
  CUDASplitInfo* cuda_best_split_info) {
  const double cnt_factor = num_data / sum_hessians;
  const bool use_l1 = lambda_l1 > 0.0f;
  const double min_gain_shift = parent_gain + min_gain_to_split;

  cuda_best_split_info->is_valid = false;

  __shared__ hist_t shared_mem_buffer[32];
  hist_t local_grad_hist = 0.0f;
  hist_t local_hess_hist = 0.0f;
  double local_gain = 0.0f;
  bool threshold_found = false;
  uint32_t threshold_value = 0;
  __shared__ uint32_t best_thread_index;
  __shared__ double shared_gain_buffer[32];
  __shared__ bool shared_found_buffer[32];
  __shared__ uint32_t shared_thread_index_buffer[32];
  const unsigned int threadIdx_x = threadIdx.x;
  const bool skip_sum = reverse ?
    (skip_default_bin && (feature_num_bin - 1 - threadIdx_x) == static_cast<int>(feature_default_bin)) :
    (skip_default_bin && (threadIdx_x + feature_mfb_offset) == static_cast<int>(feature_default_bin));
  const uint32_t feature_num_bin_minus_offset = feature_num_bin - feature_mfb_offset;
  if (!reverse) {
    if (threadIdx_x < feature_num_bin_minus_offset && !skip_sum) {
      const unsigned int bin_offset = threadIdx_x << 1;
      local_grad_hist = feature_hist_ptr[bin_offset];
      local_hess_hist = feature_hist_ptr[bin_offset + 1];
    }
  } else {
    if (threadIdx_x >= static_cast<unsigned int>(na_as_missing) &&
      threadIdx_x < feature_num_bin_minus_offset && !skip_sum) {
      const unsigned int read_index = feature_num_bin_minus_offset - 1 - threadIdx_x;
      const unsigned int bin_offset = read_index << 1;
      local_grad_hist = feature_hist_ptr[bin_offset];
      local_hess_hist = feature_hist_ptr[bin_offset + 1];
    }
  }
  __syncthreads();
  if (threadIdx_x == 0) {
    local_hess_hist += kEpsilon;
  }
  local_gain = kMinScore;
  local_grad_hist = ShufflePrefixSum(local_grad_hist, shared_mem_buffer);
  __syncthreads();
  local_hess_hist = ShufflePrefixSum(local_hess_hist, shared_mem_buffer);
  if (reverse) {
    if (threadIdx_x >= static_cast<unsigned int>(na_as_missing) && threadIdx_x <= feature_num_bin - 2 && !skip_sum) {
      const double sum_right_gradient = local_grad_hist;
      const double sum_right_hessian = local_hess_hist;
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double sum_left_gradient = sum_gradients - sum_right_gradient;
      const double sum_left_hessian = sum_hessians - sum_right_hessian;
      const data_size_t left_count = num_data - right_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          lambda_l2);
        // gain with split is worse than without split
        if (current_gain > min_gain_shift) {
          local_gain = current_gain - min_gain_shift;
          threshold_value = static_cast<uint32_t>(feature_num_bin - 2 - threadIdx_x);
          threshold_found = true;
        }
      }
    }
  } else {
    if (threadIdx_x <= feature_num_bin_minus_offset - 2 && !skip_sum) {
      const double sum_left_gradient = local_grad_hist;
      const double sum_left_hessian = local_hess_hist;
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = num_data - left_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          lambda_l2);
        // gain with split is worse than without split
        if (current_gain > min_gain_shift) {
          local_gain = current_gain - min_gain_shift;
          threshold_value = static_cast<uint32_t>(threadIdx_x + feature_mfb_offset);
          threshold_found = true;
        }
      }
    }
  }
  __syncthreads();
  const uint32_t result = ReduceBestGain(local_gain, threshold_found, threadIdx_x, shared_gain_buffer, shared_found_buffer, shared_thread_index_buffer);
  if (threadIdx_x == 0) {
    best_thread_index = result;
  }
  __syncthreads();
  if (threshold_found && threadIdx_x == best_thread_index) {
    cuda_best_split_info->is_valid = true;
    cuda_best_split_info->threshold = threshold_value;
    cuda_best_split_info->gain = local_gain;
    cuda_best_split_info->default_left = assume_out_default_left;
    if (reverse) {
      const double sum_right_gradient = local_grad_hist;
      const double sum_right_hessian = local_hess_hist - kEpsilon;
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double sum_left_gradient = sum_gradients - sum_right_gradient;
      const double sum_left_hessian = sum_hessians - sum_right_hessian - kEpsilon;
      const data_size_t left_count = num_data - right_count;
      const double left_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2);
      const double right_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2, right_output);
    } else {
      const double sum_left_gradient = local_grad_hist;
      const double sum_left_hessian = local_hess_hist - kEpsilon;
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian - kEpsilon;
      const data_size_t right_count = num_data - left_count;
      const double left_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2);
      const double right_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2, right_output);
    }
  }
}

__device__ void FindBestSplitsForLeafKernelCategoricalInner(
  // input feature information
  const hist_t* feature_hist_ptr,
  const uint32_t feature_num_bin,
  const uint8_t feature_mfb_offset,
  const uint32_t feature_default_bin,
  const int inner_feature_index,
  // input config parameter values
  const double lambda_l1,
  const double lambda_l2,
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  const double cat_smooth,
  const double cat_l2,
  const int max_cat_threshold,
  const int min_data_per_group,
  // input parent node information
  const double parent_gain,
  const double sum_gradients,
  const double sum_hessians,
  const data_size_t num_data,
  // task information
  const bool is_one_hot,
  // output parameters
  CUDASplitInfo* cuda_best_split_info) {
  __shared__ double shared_gain_buffer[32];
  __shared__ bool shared_found_buffer[32];
  __shared__ uint32_t shared_thread_index_buffer[32];
  __shared__ uint32_t best_thread_index;
  const double cnt_factor = num_data / sum_hessians;
  const bool use_l1 = lambda_l1 > 0.0f;
  const double min_gain_shift = parent_gain + min_gain_to_split;
  const double l2 = lambda_l2 + cat_l2;

  double local_gain = min_gain_shift;
  bool threshold_found = false;

  cuda_best_split_info->is_valid = false;

  const int bin_start = 1 - feature_mfb_offset;
  const int bin_end = feature_num_bin - feature_mfb_offset;
  const int threadIdx_x = static_cast<int>(threadIdx.x);
  if (is_one_hot) {
    if (threadIdx_x >= bin_start && threadIdx_x < bin_end) {
      const int bin_offset = (threadIdx_x << 1);
      const hist_t grad = feature_hist_ptr[bin_offset];
      const hist_t hess = feature_hist_ptr[bin_offset + 1];
      data_size_t cnt =
            static_cast<data_size_t>(__double2int_rn(hess * cnt_factor));
      if (cnt >= min_data_in_leaf && hess >= min_sum_hessian_in_leaf) {
        const data_size_t other_count = num_data - cnt;
        if (other_count >= min_data_in_leaf) {
          const double sum_other_hessian = sum_hessians - hess - kEpsilon;
          if (sum_other_hessian >= min_sum_hessian_in_leaf) {
            const double sum_other_gradient = sum_gradients - grad;
            double current_gain = GetSplitGains(
              sum_other_gradient, sum_other_hessian, grad,
              hess + kEpsilon, lambda_l1, use_l1,
              l2);
            if (current_gain > min_gain_shift) {
              local_gain = current_gain;
              threshold_found = true;
            }
          }
        }
      }
    }
    __syncthreads();
    const uint32_t result = ReduceBestGain(local_gain, threshold_found, threadIdx_x, shared_gain_buffer, shared_found_buffer, shared_thread_index_buffer);
    if (threadIdx_x == 0) {
      best_thread_index = result;
    }
    __syncthreads();
    if (threshold_found && threadIdx_x == best_thread_index) {
      cuda_best_split_info->is_valid = true;
      cuda_best_split_info->num_cat_threshold = 1;
      cuda_best_split_info->gain = local_gain - min_gain_shift;
      *(cuda_best_split_info->cat_threshold) = static_cast<uint32_t>(threadIdx_x + feature_mfb_offset);
      cuda_best_split_info->default_left = false;
      const int bin_offset = (threadIdx_x << 1);
      const hist_t sum_left_gradient = feature_hist_ptr[bin_offset];
      const hist_t sum_left_hessian = feature_hist_ptr[bin_offset + 1];
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double left_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, l2);
      const double right_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, l2, right_output);
    }
  } else {
    __shared__ double shared_value_buffer[NUM_THREADS_PER_BLOCK_BEST_SPLIT_FINDER];
    __shared__ int16_t shared_index_buffer[NUM_THREADS_PER_BLOCK_BEST_SPLIT_FINDER];
    __shared__ uint16_t shared_mem_buffer_uint16[32];
    __shared__ double shared_mem_buffer_double[32];
    __shared__ int used_bin;
    uint16_t is_valid_bin = 0;
    int best_dir = 0;
    double best_sum_left_gradient = 0.0f;
    double best_sum_left_hessian = 0.0f;
    if (threadIdx_x >= bin_start && threadIdx_x < bin_end) {
      const int bin_offset = (threadIdx_x << 1);
      const double hess = feature_hist_ptr[bin_offset + 1];
      if (__double2int_rn(hess * cnt_factor) >= cat_smooth) {
        const double grad = feature_hist_ptr[bin_offset];
        shared_value_buffer[threadIdx_x] = grad / (hess + cat_smooth);
        is_valid_bin = 1;
      } else {
        shared_value_buffer[threadIdx_x] = kMaxScore;
      }
    } else {
      shared_value_buffer[threadIdx_x] = kMaxScore;
    }
    shared_index_buffer[threadIdx_x] = threadIdx_x;
    __syncthreads();
    const int local_used_bin = ShuffleReduceSum<uint16_t>(is_valid_bin, shared_mem_buffer_uint16, blockDim.x);
    if (threadIdx_x == 0) {
      used_bin = local_used_bin;
    }
    __syncthreads();
    // TODO(shiyu1994): with more threads, this kernel may use out registers
    BitonicArgSort_1024<double, int16_t, true>(shared_value_buffer, shared_index_buffer, bin_end);
    __syncthreads();
    const int max_num_cat = min(max_cat_threshold, (used_bin + 1) / 2);

    // left to right
    double grad = 0.0f;
    double hess = 0.0f;
    if (threadIdx_x < used_bin && threadIdx_x < max_num_cat) {
      const int bin_offset = (shared_index_buffer[threadIdx_x] << 1);
      grad = feature_hist_ptr[bin_offset];
      hess = feature_hist_ptr[bin_offset + 1];
    }
    if (threadIdx_x == 0) {
      hess += kEpsilon;
    }
    __syncthreads();
    double sum_left_gradient = ShufflePrefixSum<double>(grad, shared_mem_buffer_double);
    __syncthreads();
    double sum_left_hessian = ShufflePrefixSum<double>(hess, shared_mem_buffer_double);
    // TODO(shiyu1994): constrain the split with min_data_in_group
    if (threadIdx_x < used_bin && threadIdx_x < max_num_cat) {
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = num_data - left_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          l2);
        // gain with split is worse than without split
        if (current_gain > local_gain) {
          local_gain = current_gain;
          threshold_found = true;
          best_dir = 1;
          best_sum_left_gradient = sum_left_gradient;
          best_sum_left_hessian = sum_left_hessian;
        }
      }
    }
    __syncthreads();

    // right to left
    grad = 0.0f;
    hess = 0.0f;
    if (threadIdx_x < used_bin && threadIdx_x < max_num_cat) {
      const int bin_offset = (shared_index_buffer[used_bin - 1 - threadIdx_x] << 1);
      grad = feature_hist_ptr[bin_offset];
      hess = feature_hist_ptr[bin_offset + 1];
    }
    if (threadIdx_x == 0) {
      hess += kEpsilon;
    }
    __syncthreads();
    sum_left_gradient = ShufflePrefixSum<double>(grad, shared_mem_buffer_double);
    __syncthreads();
    sum_left_hessian = ShufflePrefixSum<double>(hess, shared_mem_buffer_double);
    // TODO(shiyu1994): constrain the split with min_data_in_group
    if (threadIdx_x < used_bin && threadIdx_x < max_num_cat) {
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = num_data - left_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          l2);
        // gain with split is worse than without split
        if (current_gain > local_gain) {
          local_gain = current_gain;
          threshold_found = true;
          best_dir = -1;
          best_sum_left_gradient = sum_left_gradient;
          best_sum_left_hessian = sum_left_hessian;
        }
      }
    }
    __syncthreads();

    const uint32_t result = ReduceBestGain(local_gain, threshold_found, threadIdx_x, shared_gain_buffer, shared_found_buffer, shared_thread_index_buffer);
    if (threadIdx_x == 0) {
      best_thread_index = result;
    }
    __syncthreads();
    if (threshold_found && threadIdx_x == best_thread_index) {
      cuda_best_split_info->is_valid = true;
      cuda_best_split_info->num_cat_threshold = threadIdx_x + 1;
      cuda_best_split_info->gain = local_gain - min_gain_shift;
      if (best_dir == 1) {
        for (int i = 0; i < threadIdx_x + 1; ++i) {
          (cuda_best_split_info->cat_threshold)[i] = shared_index_buffer[i] + feature_mfb_offset;
        }
      } else {
        for (int i = 0; i < threadIdx_x + 1; ++i) {
          (cuda_best_split_info->cat_threshold)[i] = shared_index_buffer[used_bin - 1 - i] + feature_mfb_offset;
        }
      }
      cuda_best_split_info->default_left = false;
      const hist_t sum_left_gradient = best_sum_left_gradient;
      const hist_t sum_left_hessian = best_sum_left_hessian;
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double left_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, l2);
      const double right_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, l2, right_output);
    }
  }
}

__global__ void FindBestSplitsForLeafKernel(
  // input feature information
  const uint32_t* feature_hist_offsets,
  const uint8_t* feature_mfb_offsets,
  const uint32_t* feature_default_bins,
  const uint32_t* feature_num_bins,
  const int8_t* is_feature_used_bytree,
  const int8_t* is_categorical,
  // input task information
  const bool larger_only,
  const int num_tasks,
  const int* task_feature_index,
  const uint8_t* task_reverse,
  const uint8_t* task_skip_default_bin,
  const uint8_t* task_na_as_missing,
  const uint8_t* task_out_default_left,
  // input leaf information
  const int smaller_leaf_index,
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const int larger_leaf_index,
  const CUDALeafSplitsStruct* larger_leaf_splits,
  // input config parameter values
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  const double lambda_l1,
  const double lambda_l2,
  const double cat_smooth,
  const double cat_l2,
  const int max_cat_threshold,
  const int min_data_per_group,
  const int max_cat_to_onehot,
  // output
  CUDASplitInfo* cuda_best_split_info) {
  const unsigned int task_index = blockIdx.x % num_tasks;
  const bool is_larger = static_cast<bool>(blockIdx.x >= num_tasks || larger_only);
  const int inner_feature_index = task_feature_index[task_index];
  const double parent_gain = is_larger ? larger_leaf_splits->gain : smaller_leaf_splits->gain;
  const double sum_gradients = is_larger ? larger_leaf_splits->sum_of_gradients : smaller_leaf_splits->sum_of_gradients;
  const double sum_hessians = (is_larger ? larger_leaf_splits->sum_of_hessians : smaller_leaf_splits->sum_of_hessians) + 2 * kEpsilon;
  const double num_data = is_larger ? larger_leaf_splits->num_data_in_leaf : smaller_leaf_splits->num_data_in_leaf;
  const unsigned int output_offset = is_larger ? (task_index + num_tasks) : task_index;
  CUDASplitInfo* out = cuda_best_split_info + output_offset;
  if (is_feature_used_bytree[inner_feature_index]) {
    const hist_t* hist_ptr = (is_larger ? larger_leaf_splits->hist_in_leaf : smaller_leaf_splits->hist_in_leaf) + feature_hist_offsets[inner_feature_index] * 2;
    if (is_categorical[inner_feature_index]) {
      const bool is_one_hot = feature_num_bins[inner_feature_index] <= max_cat_to_onehot;
      FindBestSplitsForLeafKernelCategoricalInner(
        // input feature information
        hist_ptr,
        feature_num_bins[inner_feature_index],
        feature_mfb_offsets[inner_feature_index],
        feature_default_bins[inner_feature_index],
        inner_feature_index,
        // input config parameter values
        lambda_l1,
        lambda_l2,
        min_data_in_leaf,
        min_sum_hessian_in_leaf,
        min_gain_to_split,
        cat_smooth,
        cat_l2,
        max_cat_threshold,
        min_data_per_group,
        // input parent node information
        parent_gain,
        sum_gradients,
        sum_hessians,
        num_data,
        // input task information
        is_one_hot,
        // output parameters
        out);
    } else {
      const bool reverse = static_cast<bool>(task_reverse[task_index]);
      const bool skip_default_bin = static_cast<bool>(task_skip_default_bin[task_index]);
      const bool na_as_missing = static_cast<bool>(task_na_as_missing[task_index]);
      const bool assume_out_default_left = task_out_default_left[task_index];
      FindBestSplitsForLeafKernelInner(
        // input feature information
        hist_ptr,
        feature_num_bins[inner_feature_index],
        feature_mfb_offsets[inner_feature_index],
        feature_default_bins[inner_feature_index],
        inner_feature_index,
        // input config parameter values
        lambda_l1,
        lambda_l2,
        min_data_in_leaf,
        min_sum_hessian_in_leaf,
        min_gain_to_split,
        // input parent node information
        parent_gain,
        sum_gradients,
        sum_hessians,
        num_data,
        // input task information
        reverse,
        skip_default_bin,
        na_as_missing,
        assume_out_default_left,
        // output parameters
        out);
    }
  } else {
    out->is_valid = false;
  }
}

__device__ void FindBestSplitsForLeafKernelInner_GlobalMemory(
  // input feature information
  const hist_t* feature_hist_ptr,
  const uint32_t feature_num_bin,
  const uint8_t feature_mfb_offset,
  const uint32_t feature_default_bin,
  const int inner_feature_index,
  // input config parameter values
  const double lambda_l1,
  const double lambda_l2,
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  // input parent node information
  const double parent_gain,
  const double sum_gradients,
  const double sum_hessians,
  const data_size_t num_data,
  // input task information
  const bool reverse,
  const bool skip_default_bin,
  const bool na_as_missing,
  const uint8_t assume_out_default_left,
  // buffer
  hist_t* hist_grad_buffer_ptr,
  hist_t* hist_hess_buffer_ptr,
  // output parameters
  CUDASplitInfo* cuda_best_split_info) {
  const double cnt_factor = num_data / sum_hessians;
  const bool use_l1 = lambda_l1 > 0.0f;
  const double min_gain_shift = parent_gain + min_gain_to_split;

  cuda_best_split_info->is_valid = false;
  double local_gain = 0.0f;
  bool threshold_found = false;
  uint32_t threshold_value = 0;
  __shared__ uint32_t best_thread_index;
  __shared__ double shared_gain_buffer[32];
  __shared__ bool shared_found_buffer[32];
  __shared__ uint32_t shared_thread_index_buffer[32];
  const unsigned int threadIdx_x = threadIdx.x;
  const uint32_t feature_num_bin_minus_offset = feature_num_bin - feature_mfb_offset;
  if (!reverse) {
    for (unsigned int bin = threadIdx_x; bin < feature_num_bin_minus_offset; ++bin) {
      const bool skip_sum =
        (skip_default_bin && (bin + feature_mfb_offset) == static_cast<int>(feature_default_bin));
      if (!skip_sum) {
        const unsigned int bin_offset = bin << 1;
        hist_grad_buffer_ptr[bin] = feature_hist_ptr[bin_offset];
        hist_hess_buffer_ptr[bin] = feature_hist_ptr[bin_offset + 1];
      } else {
        hist_grad_buffer_ptr[bin] = 0.0f;
        hist_hess_buffer_ptr[bin] = 0.0f;
      }
    }
  } else {
    for (unsigned int bin = threadIdx_x; bin < feature_num_bin_minus_offset; ++bin) {
      const bool skip_sum = bin >= static_cast<unsigned int>(na_as_missing) && 
        (skip_default_bin && (feature_num_bin - 1 - bin) == static_cast<int>(feature_default_bin));
      if (!skip_sum) {
        const unsigned int read_index = feature_num_bin_minus_offset - 1 - bin;
        const unsigned int bin_offset = read_index << 1;
        hist_grad_buffer_ptr[bin] = feature_hist_ptr[bin_offset];
        hist_hess_buffer_ptr[bin] = feature_hist_ptr[bin_offset + 1];
      } else {
        hist_grad_buffer_ptr[bin] = 0.0f;
        hist_hess_buffer_ptr[bin] = 0.0f;
      }
    }
  }
  __syncthreads();
  if (threadIdx_x == 0) {
    hist_hess_buffer_ptr[0] += kEpsilon;
  }
  local_gain = kMinScore;
  GlobalMemoryPrefixSum(hist_grad_buffer_ptr, static_cast<size_t>(feature_num_bin_minus_offset));
  __syncthreads();
  GlobalMemoryPrefixSum(hist_hess_buffer_ptr, static_cast<size_t>(feature_num_bin_minus_offset));
  if (reverse) {
    for (unsigned int bin = threadIdx_x; bin < feature_num_bin_minus_offset; ++bin) {
      const bool skip_sum = (bin >= static_cast<unsigned int>(na_as_missing) &&
        (skip_default_bin && (feature_num_bin - 1 - bin) == static_cast<int>(feature_default_bin)));
      if (!skip_sum) {
        const double sum_right_gradient = hist_grad_buffer_ptr[bin];
        const double sum_right_hessian = hist_hess_buffer_ptr[bin];
        const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
        const double sum_left_gradient = sum_gradients - sum_right_gradient;
        const double sum_left_hessian = sum_hessians - sum_right_hessian;
        const data_size_t left_count = num_data - right_count;
        if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
          sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
          double current_gain = GetSplitGains(
            sum_left_gradient, sum_left_hessian, sum_right_gradient,
            sum_right_hessian, lambda_l1, use_l1,
            lambda_l2);
          // gain with split is worse than without split
          if (current_gain > min_gain_shift) {
            local_gain = current_gain - min_gain_shift;
            threshold_value = static_cast<uint32_t>(feature_num_bin - 2 - bin);
            threshold_found = true;
          }
        }
      }
    }
  } else {
    for (unsigned int bin = threadIdx_x; bin <= feature_num_bin_minus_offset - 2; ++bin) {
      const bool skip_sum =
        (skip_default_bin && (bin + feature_mfb_offset) == static_cast<int>(feature_default_bin));
      if (!skip_sum) {
        const double sum_left_gradient = hist_grad_buffer_ptr[bin];
        const double sum_left_hessian = hist_hess_buffer_ptr[bin];
        const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
        const double sum_right_gradient = sum_gradients - sum_left_gradient;
        const double sum_right_hessian = sum_hessians - sum_left_hessian;
        const data_size_t right_count = num_data - left_count;
        if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
          sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
          double current_gain = GetSplitGains(
            sum_left_gradient, sum_left_hessian, sum_right_gradient,
            sum_right_hessian, lambda_l1, use_l1,
            lambda_l2);
          // gain with split is worse than without split
          if (current_gain > min_gain_shift) {
            local_gain = current_gain - min_gain_shift;
            threshold_value = static_cast<uint32_t>(bin + feature_mfb_offset);
            threshold_found = true;
          }
        }
      }
    }
  }
  __syncthreads();
  const uint32_t result = ReduceBestGain(local_gain, threshold_found, threadIdx_x, shared_gain_buffer, shared_found_buffer, shared_thread_index_buffer);
  if (threadIdx_x == 0) {
    best_thread_index = result;
  }
  __syncthreads();
  if (threshold_found && threadIdx_x == best_thread_index) {
    cuda_best_split_info->is_valid = true;
    cuda_best_split_info->threshold = threshold_value;
    cuda_best_split_info->gain = local_gain;
    cuda_best_split_info->default_left = assume_out_default_left;
    if (reverse) {
      const unsigned int best_bin = static_cast<uint32_t>(feature_num_bin - 2 - threshold_value);
      const double sum_right_gradient = hist_grad_buffer_ptr[best_bin];
      const double sum_right_hessian = hist_hess_buffer_ptr[best_bin] - kEpsilon;
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double sum_left_gradient = sum_gradients - sum_right_gradient;
      const double sum_left_hessian = sum_hessians - sum_right_hessian - kEpsilon;
      const data_size_t left_count = num_data - right_count;
      const double left_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2);
      const double right_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2, right_output);
    } else {
      const unsigned int best_bin = static_cast<uint32_t>(threshold_value - feature_mfb_offset);
      const double sum_left_gradient = hist_grad_buffer_ptr[best_bin];
      const double sum_left_hessian = hist_hess_buffer_ptr[best_bin] - kEpsilon;
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian - kEpsilon;
      const data_size_t right_count = num_data - left_count;
      const double left_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2);
      const double right_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2, right_output);
    }
  }
}

__device__ void FindBestSplitsForLeafKernelCategoricalInner_GlobalMemory(
  // input feature information
  const hist_t* feature_hist_ptr,
  const uint32_t feature_num_bin,
  const uint8_t feature_mfb_offset,
  const uint32_t feature_default_bin,
  const int inner_feature_index,
  // input config parameter values
  const double lambda_l1,
  const double lambda_l2,
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  const double cat_smooth,
  const double cat_l2,
  const int max_cat_threshold,
  const int min_data_per_group,
  // input parent node information
  const double parent_gain,
  const double sum_gradients,
  const double sum_hessians,
  const data_size_t num_data,
  // task information
  const bool is_one_hot,
  // buffer
  hist_t* hist_grad_buffer_ptr,
  hist_t* hist_hess_buffer_ptr,
  hist_t* hist_stat_buffer_ptr,
  data_size_t* hist_index_buffer_ptr,
  // output parameters
  CUDASplitInfo* cuda_best_split_info) {
  __shared__ double shared_gain_buffer[32];
  __shared__ bool shared_found_buffer[32];
  __shared__ uint32_t shared_thread_index_buffer[32];
  __shared__ uint32_t best_thread_index;
  const double cnt_factor = num_data / sum_hessians;
  const bool use_l1 = lambda_l1 > 0.0f;
  const double min_gain_shift = parent_gain + min_gain_to_split;
  const double l2 = lambda_l2 + cat_l2;

  double local_gain = kMinScore;
  bool threshold_found = false;

  cuda_best_split_info->is_valid = false;

  const int bin_start = 1 - feature_mfb_offset;
  const int bin_end = feature_num_bin - feature_mfb_offset;
  int best_threshold = -1;
  const int threadIdx_x = static_cast<int>(threadIdx.x);
  if (is_one_hot) {
    for (int bin = bin_start + threadIdx_x; bin < bin_end; bin += static_cast<int>(blockDim.x)) {
      const int bin_offset = (bin << 1);
      const hist_t grad = feature_hist_ptr[bin_offset];
      const hist_t hess = feature_hist_ptr[bin_offset + 1];
      data_size_t cnt =
            static_cast<data_size_t>(__double2int_rn(hess * cnt_factor));
      if (cnt >= min_data_in_leaf && hess >= min_sum_hessian_in_leaf) {
        const data_size_t other_count = num_data - cnt;
        if (other_count >= min_data_in_leaf) {
          const double sum_other_hessian = sum_hessians - hess - kEpsilon;
          if (sum_other_hessian >= min_sum_hessian_in_leaf) {
            const double sum_other_gradient = sum_gradients - grad;
            double current_gain = GetSplitGains(
              sum_other_gradient, sum_other_hessian, grad,
              hess + kEpsilon, lambda_l1, use_l1,
              l2);
            if (current_gain > min_gain_shift) {
              best_threshold = bin;
              local_gain = current_gain - min_gain_shift;
              threshold_found = true;
            }
          }
        }
      }
    }
    __syncthreads();
    const uint32_t result = ReduceBestGain(local_gain, threshold_found, threadIdx_x, shared_gain_buffer, shared_found_buffer, shared_thread_index_buffer);
    if (threadIdx_x == 0) {
      best_thread_index = result;
    }
    __syncthreads();
    if (threshold_found && threadIdx_x == best_thread_index) {
      cuda_best_split_info->is_valid = true;
      cuda_best_split_info->num_cat_threshold = 1;
      cuda_best_split_info->cat_threshold = new uint32_t[1];
      *(cuda_best_split_info->cat_threshold) = static_cast<uint32_t>(best_threshold);
      cuda_best_split_info->default_left = false;
      const int bin_offset = (best_threshold << 1);
      const hist_t sum_left_gradient = feature_hist_ptr[bin_offset];
      const hist_t sum_left_hessian = feature_hist_ptr[bin_offset + 1];
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double left_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, l2);
      const double right_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, l2, right_output);
    }
  } else {
    __shared__ uint16_t shared_mem_buffer_uint16[32];
    __shared__ int used_bin;
    uint16_t is_valid_bin = 0;
    int best_dir = 0;
    double best_sum_left_gradient = 0.0f;
    double best_sum_left_hessian = 0.0f;
    for (int bin = 0; bin < bin_end; bin += static_cast<int>(blockDim.x)) {
      if (bin >= bin_start) {
        const int bin_offset = (bin << 1);
        const double hess = feature_hist_ptr[bin_offset + 1];
        if (__double2int_rn(hess * cnt_factor) >= cat_smooth) {
          const double grad = feature_hist_ptr[bin_offset];
          hist_stat_buffer_ptr[bin] = grad / (hess + cat_smooth);
          hist_index_buffer_ptr[bin] = threadIdx_x;
          is_valid_bin = 1;
        } else {
          hist_stat_buffer_ptr[bin] = kMaxScore;
          hist_index_buffer_ptr[bin] = -1;
        }
      }
    }
    __syncthreads();
    const int local_used_bin = ShuffleReduceSum<uint16_t>(is_valid_bin, shared_mem_buffer_uint16, blockDim.x);
    if (threadIdx_x == 0) {
      used_bin = local_used_bin;
    }
    __syncthreads();
    BitonicArgSortDevice<double, data_size_t, true, NUM_THREADS_PER_BLOCK_BEST_SPLIT_FINDER, 11>(
      hist_stat_buffer_ptr, hist_index_buffer_ptr, feature_num_bin - feature_mfb_offset);
    const int max_num_cat = min(max_cat_threshold, (used_bin + 1) / 2);
    __syncthreads();

    // left to right
    if (threadIdx_x < used_bin && threadIdx_x < max_num_cat) {
      const int bin_offset = (hist_index_buffer_ptr[threadIdx_x] << 1);
      hist_grad_buffer_ptr[threadIdx_x] = feature_hist_ptr[bin_offset];
      hist_hess_buffer_ptr[threadIdx_x] = feature_hist_ptr[bin_offset + 1];
    }
    if (threadIdx_x == 0) {
      hist_hess_buffer_ptr[0] += kEpsilon;
    }
    __syncthreads();
    GlobalMemoryPrefixSum<double>(hist_grad_buffer_ptr, static_cast<size_t>(bin_end));
    __syncthreads();
    GlobalMemoryPrefixSum<double>(hist_hess_buffer_ptr, static_cast<size_t>(bin_end));
    // TODO(shiyu1994): constrain the split with min_data_in_group
    if (threadIdx_x < used_bin && threadIdx_x < max_num_cat) {
      const double sum_left_gradient = hist_grad_buffer_ptr[threadIdx_x];
      const double sum_left_hessian = hist_hess_buffer_ptr[threadIdx_x];
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = num_data - left_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          l2);
        // gain with split is worse than without split
        if (current_gain > min_gain_shift) {
          local_gain = current_gain - min_gain_shift;
          threshold_found = true;
          best_dir = 1;
          best_sum_left_gradient = sum_left_gradient;
          best_sum_left_hessian = sum_left_hessian;
        }
      }
    }
    __syncthreads();

    // right to left
    if (threadIdx_x < used_bin && threadIdx_x < max_num_cat) {
      const int bin_offset = (hist_index_buffer_ptr[used_bin - 1 - threadIdx_x] << 1);
      hist_grad_buffer_ptr[threadIdx_x] = feature_hist_ptr[bin_offset];
      hist_hess_buffer_ptr[threadIdx_x] = feature_hist_ptr[bin_offset + 1];
    }
    if (threadIdx_x == 0) {
      hist_hess_buffer_ptr[0] += kEpsilon;
    }
    __syncthreads();
    GlobalMemoryPrefixSum<double>(hist_grad_buffer_ptr, static_cast<size_t>(bin_end));
    __syncthreads();
    GlobalMemoryPrefixSum<double>(hist_hess_buffer_ptr, static_cast<size_t>(bin_end));
    // TODO(shiyu1994): constrain the split with min_data_in_group
    if (threadIdx_x < used_bin && threadIdx_x < max_num_cat) {
      const double sum_left_gradient = hist_grad_buffer_ptr[threadIdx_x];
      const double sum_left_hessian = hist_hess_buffer_ptr[threadIdx_x];
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = num_data - left_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          l2);
        // gain with split is worse than without split
        if (current_gain > min_gain_shift) {
          local_gain = current_gain - min_gain_shift;
          threshold_found = true;
          best_dir = -1;
          best_sum_left_gradient = sum_left_gradient;
          best_sum_left_hessian = sum_left_hessian;
        }
      }
    }
    __syncthreads();

    const uint32_t result = ReduceBestGain(local_gain, threshold_found, threadIdx_x, shared_gain_buffer, shared_found_buffer, shared_thread_index_buffer);
    if (threadIdx_x == 0) {
      best_thread_index = result;
    }
    __syncthreads();
    if (threshold_found && threadIdx_x == best_thread_index) {
      cuda_best_split_info->is_valid = true;
      cuda_best_split_info->num_cat_threshold = threadIdx_x + 1;
      cuda_best_split_info->cat_threshold = new uint32_t[threadIdx_x + 1];
      cuda_best_split_info->gain = local_gain;
      if (best_dir == 1) {
        for (int i = 0; i < threadIdx_x + 1; ++i) {
          (cuda_best_split_info->cat_threshold)[i] = hist_index_buffer_ptr[i] + feature_mfb_offset;
        }
      } else {
        for (int i = 0; i < threadIdx_x + 1; ++i) {
          (cuda_best_split_info->cat_threshold)[i] = hist_index_buffer_ptr[used_bin - 1 - i] + feature_mfb_offset;
        }
      }
      cuda_best_split_info->default_left = false;
      const hist_t sum_left_gradient = best_sum_left_gradient;
      const hist_t sum_left_hessian = best_sum_left_hessian;
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double left_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, l2);
      const double right_output = CUDABestSplitFinder::CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, l2, right_output);
    }
  }
}

__global__ void FindBestSplitsForLeafKernel_GlobalMemory(
  // input feature information
  const uint32_t* feature_hist_offsets,
  const uint8_t* feature_mfb_offsets,
  const uint32_t* feature_default_bins,
  const uint32_t* feature_num_bins,
  const int8_t* is_feature_used_bytree,
  const int8_t* is_categorical,
  // input task information
  const bool larger_only,
  const int num_tasks,
  const int* task_feature_index,
  const uint8_t* task_reverse,
  const uint8_t* task_skip_default_bin,
  const uint8_t* task_na_as_missing,
  const uint8_t* task_out_default_left,
  // input leaf information
  const int smaller_leaf_index,
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const int larger_leaf_index,
  const CUDALeafSplitsStruct* larger_leaf_splits,
  // input config parameter values
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  const double lambda_l1,
  const double lambda_l2,
  const double cat_smooth,
  const double cat_l2,
  const int max_cat_threshold,
  const int min_data_per_group,
  const int max_cat_to_onehot,
  // buffer
  hist_t* feature_hist_grad_buffer,
  hist_t* feature_hist_hess_buffer,
  hist_t* feature_hist_stat_buffer,
  data_size_t* feature_hist_index_buffer,
  // output
  CUDASplitInfo* cuda_best_split_info) {
  const unsigned int task_index = blockIdx.x % num_tasks;
  const bool is_larger = static_cast<bool>(blockIdx.x >= num_tasks || larger_only);
  const int inner_feature_index = task_feature_index[task_index];
  const double parent_gain = is_larger ? larger_leaf_splits->gain : smaller_leaf_splits->gain;
  const double sum_gradients = is_larger ? larger_leaf_splits->sum_of_gradients : smaller_leaf_splits->sum_of_gradients;
  const double sum_hessians = (is_larger ? larger_leaf_splits->sum_of_hessians : smaller_leaf_splits->sum_of_hessians) + 2 * kEpsilon;
  const double num_data = is_larger ? larger_leaf_splits->num_data_in_leaf : smaller_leaf_splits->num_data_in_leaf;
  const unsigned int output_offset = is_larger ? (task_index + num_tasks) : task_index;
  CUDASplitInfo* out = cuda_best_split_info + output_offset;
  if (is_feature_used_bytree[inner_feature_index]) {
    const hist_t* hist_ptr = (is_larger ? larger_leaf_splits->hist_in_leaf : smaller_leaf_splits->hist_in_leaf) + feature_hist_offsets[inner_feature_index] * 2;
    hist_t* hist_grad_buffer_ptr = feature_hist_grad_buffer + feature_hist_offsets[inner_feature_index] * 2;
    hist_t* hist_hess_buffer_ptr = feature_hist_hess_buffer + feature_hist_offsets[inner_feature_index] * 2;
    hist_t* hist_stat_buffer_ptr = feature_hist_stat_buffer + feature_hist_offsets[inner_feature_index] * 2;
    data_size_t* hist_index_buffer_ptr = feature_hist_index_buffer + feature_hist_offsets[inner_feature_index] * 2;
    if (is_categorical[inner_feature_index]) {
      const bool is_one_hot = feature_num_bins[inner_feature_index] <= max_cat_to_onehot;
      FindBestSplitsForLeafKernelCategoricalInner_GlobalMemory(
        // input feature information
        hist_ptr,
        feature_num_bins[inner_feature_index],
        feature_mfb_offsets[inner_feature_index],
        feature_default_bins[inner_feature_index],
        inner_feature_index,
        // input config parameter values
        lambda_l1,
        lambda_l2,
        min_data_in_leaf,
        min_sum_hessian_in_leaf,
        min_gain_to_split,
        cat_smooth,
        cat_l2,
        max_cat_threshold,
        min_data_per_group,
        // input parent node information
        parent_gain,
        sum_gradients,
        sum_hessians,
        num_data,
        // input task information
        is_one_hot,
        // buffer
        hist_grad_buffer_ptr,
        hist_hess_buffer_ptr,
        hist_stat_buffer_ptr,
        hist_index_buffer_ptr,
        // output parameters
        out);
    } else {
      const bool reverse = static_cast<bool>(task_reverse[task_index]);
      const bool skip_default_bin = static_cast<bool>(task_skip_default_bin[task_index]);
      const bool na_as_missing = static_cast<bool>(task_na_as_missing[task_index]);
      const bool assume_out_default_left = task_out_default_left[task_index];
      FindBestSplitsForLeafKernelInner_GlobalMemory(
        // input feature information
        hist_ptr,
        feature_num_bins[inner_feature_index],
        feature_mfb_offsets[inner_feature_index],
        feature_default_bins[inner_feature_index],
        inner_feature_index,
        // input config parameter values
        lambda_l1,
        lambda_l2,
        min_data_in_leaf,
        min_sum_hessian_in_leaf,
        min_gain_to_split,
        // input parent node information
        parent_gain,
        sum_gradients,
        sum_hessians,
        num_data,
        // input task information
        reverse,
        skip_default_bin,
        na_as_missing,
        assume_out_default_left,
        // buffer
        hist_grad_buffer_ptr,
        hist_hess_buffer_ptr,
        // output parameters
        out);
      }
  } else {
    out->is_valid = false;
  }
}

void CUDABestSplitFinder::LaunchFindBestSplitsForLeafKernel(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const CUDALeafSplitsStruct* larger_leaf_splits,
  const int smaller_leaf_index,
  const int larger_leaf_index,
  const bool is_smaller_leaf_valid,
  const bool is_larger_leaf_valid) {
  if (!is_smaller_leaf_valid && !is_larger_leaf_valid) {
    return;
  }
  bool larger_only = false;
  if (!is_smaller_leaf_valid) {
    larger_only = true;
  }
  if (!use_global_memory_) {
    if (!larger_only) {
      FindBestSplitsForLeafKernel<<<num_tasks_, NUM_THREADS_PER_BLOCK_BEST_SPLIT_FINDER, 0, cuda_streams_[0]>>>(
        // input feature information
        cuda_feature_hist_offsets_,
        cuda_feature_mfb_offsets_,
        cuda_feature_default_bins_,
        cuda_feature_num_bins_,
        cuda_is_feature_used_bytree_,
        cuda_is_categorical_,
        // input task information
        larger_only,
        num_tasks_,
        cuda_task_feature_index_,
        cuda_task_reverse_,
        cuda_task_skip_default_bin_,
        cuda_task_na_as_missing_,
        cuda_task_out_default_left_,
        // input leaf information
        smaller_leaf_index,
        smaller_leaf_splits,
        larger_leaf_index,
        larger_leaf_splits,
        // configuration parameter values
        min_data_in_leaf_,
        min_sum_hessian_in_leaf_,
        min_gain_to_split_,
        lambda_l1_,
        lambda_l2_,
        cat_smooth_,
        cat_l2_,
        max_cat_threshold_,
        min_data_per_group_,
        max_cat_to_onehot_,
        // output parameters
        cuda_best_split_info_);
    }
    SynchronizeCUDADevice(__FILE__, __LINE__);
    if (larger_leaf_index >= 0) {
      FindBestSplitsForLeafKernel<<<num_tasks_, NUM_THREADS_PER_BLOCK_BEST_SPLIT_FINDER, 0, cuda_streams_[1]>>>(
        // input feature information
        cuda_feature_hist_offsets_,
        cuda_feature_mfb_offsets_,
        cuda_feature_default_bins_,
        cuda_feature_num_bins_,
        cuda_is_feature_used_bytree_,
        cuda_is_categorical_,
        // input task information
        true,
        num_tasks_,
        cuda_task_feature_index_,
        cuda_task_reverse_,
        cuda_task_skip_default_bin_,
        cuda_task_na_as_missing_,
        cuda_task_out_default_left_,
        // input leaf information
        smaller_leaf_index,
        smaller_leaf_splits,
        larger_leaf_index,
        larger_leaf_splits,
        // configuration parameter values
        min_data_in_leaf_,
        min_sum_hessian_in_leaf_,
        min_gain_to_split_,
        lambda_l1_,
        lambda_l2_,
        cat_smooth_,
        cat_l2_,
        max_cat_threshold_,
        min_data_per_group_,
        max_cat_to_onehot_,
        // output parameters
        cuda_best_split_info_);
    }
  } else {
    if (!larger_only) {
      FindBestSplitsForLeafKernel_GlobalMemory<<<num_tasks_, NUM_THREADS_PER_BLOCK_BEST_SPLIT_FINDER, 0, cuda_streams_[0]>>>(
        // input feature information
        cuda_feature_hist_offsets_,
        cuda_feature_mfb_offsets_,
        cuda_feature_default_bins_,
        cuda_feature_num_bins_,
        cuda_is_feature_used_bytree_,
        cuda_is_categorical_,
        // input task information
        larger_only,
        num_tasks_,
        cuda_task_feature_index_,
        cuda_task_reverse_,
        cuda_task_skip_default_bin_,
        cuda_task_na_as_missing_,
        cuda_task_out_default_left_,
        // input leaf information
        smaller_leaf_index,
        smaller_leaf_splits,
        larger_leaf_index,
        larger_leaf_splits,
        // configuration parameter values
        min_data_in_leaf_,
        min_sum_hessian_in_leaf_,
        min_gain_to_split_,
        lambda_l1_,
        lambda_l2_,
        cat_smooth_,
        cat_l2_,
        max_cat_threshold_,
        min_data_per_group_,
        max_cat_to_onehot_,
        // buffer
        cuda_feature_hist_grad_buffer_,
        cuda_feature_hist_hess_buffer_,
        cuda_feature_hist_stat_buffer_,
        cuda_feature_hist_index_buffer_,
        // output parameters
        cuda_best_split_info_);
    }
    SynchronizeCUDADevice(__FILE__, __LINE__);
    if (larger_leaf_index >= 0) {
      FindBestSplitsForLeafKernel_GlobalMemory<<<num_tasks_, NUM_THREADS_PER_BLOCK_BEST_SPLIT_FINDER, 0, cuda_streams_[1]>>>(
        // input feature information
        cuda_feature_hist_offsets_,
        cuda_feature_mfb_offsets_,
        cuda_feature_default_bins_,
        cuda_feature_num_bins_,
        cuda_is_feature_used_bytree_,
        cuda_is_categorical_,
        // input task information
        true,
        num_tasks_,
        cuda_task_feature_index_,
        cuda_task_reverse_,
        cuda_task_skip_default_bin_,
        cuda_task_na_as_missing_,
        cuda_task_out_default_left_,
        // input leaf information
        smaller_leaf_index,
        smaller_leaf_splits,
        larger_leaf_index,
        larger_leaf_splits,
        // configuration parameter values
        min_data_in_leaf_,
        min_sum_hessian_in_leaf_,
        min_gain_to_split_,
        lambda_l1_,
        lambda_l2_,
        cat_smooth_,
        cat_l2_,
        max_cat_threshold_,
        min_data_per_group_,
        max_cat_to_onehot_,
        // buffer
        cuda_feature_hist_grad_buffer_,
        cuda_feature_hist_hess_buffer_,
        cuda_feature_hist_stat_buffer_,
        cuda_feature_hist_index_buffer_,
        // output parameters
        cuda_best_split_info_);
    }
  }
}

__device__ void ReduceBestSplit(bool* found, double* gain, uint32_t* shared_read_index,
  uint32_t num_features_aligned) {
  const uint32_t threadIdx_x = threadIdx.x;
  for (unsigned int s = 1; s < num_features_aligned; s <<= 1) {
    if (threadIdx_x % (2 * s) == 0 && (threadIdx_x + s) < num_features_aligned) {
      const uint32_t pos_to_compare = threadIdx_x + s;
      if ((!found[threadIdx_x] && found[pos_to_compare]) ||
        (found[threadIdx_x] && found[pos_to_compare] && gain[threadIdx_x] < gain[pos_to_compare])) {
        found[threadIdx_x] = found[pos_to_compare];
        gain[threadIdx_x] = gain[pos_to_compare];
        shared_read_index[threadIdx_x] = shared_read_index[pos_to_compare];
      }
    }
    __syncthreads();
  }
}

__global__ void SyncBestSplitForLeafKernel(const int smaller_leaf_index, const int larger_leaf_index,
  CUDASplitInfo* cuda_leaf_best_split_info,
  // input parameters
  const int* cuda_task_feature_index,
  const CUDASplitInfo* cuda_best_split_info,
  const uint32_t* cuda_feature_default_bins,
  const int num_tasks,
  const int num_tasks_aligned,
  const int num_blocks_per_leaf,
  const bool larger_only,
  const int num_leaves) {
  __shared__ double shared_gain_buffer[32];
  __shared__ bool shared_found_buffer[32];
  __shared__ uint32_t shared_thread_index_buffer[32];
  const uint32_t threadIdx_x = threadIdx.x;
  const uint32_t blockIdx_x = blockIdx.x;

  bool best_found = false;
  double best_gain = kMinScore;
  uint32_t shared_read_index = 0;

  const bool is_smaller = (blockIdx_x < static_cast<unsigned int>(num_blocks_per_leaf) && !larger_only);
  const uint32_t leaf_block_index = (is_smaller || larger_only) ? blockIdx_x : (blockIdx_x - static_cast<unsigned int>(num_blocks_per_leaf));
  const int task_index = static_cast<int>(leaf_block_index * blockDim.x + threadIdx_x);
  const uint32_t read_index = is_smaller ? static_cast<uint32_t>(task_index) : static_cast<uint32_t>(task_index + num_tasks);
  if (task_index < num_tasks) {
    best_found = cuda_best_split_info[read_index].is_valid;
    best_gain = cuda_best_split_info[read_index].gain;
    shared_read_index = read_index;
  } else {
    best_found = false;
  }

  __syncthreads();
  const uint32_t best_read_index = ReduceBestGain(best_gain, best_found, shared_read_index,
      shared_gain_buffer, shared_found_buffer, shared_thread_index_buffer);
  if (threadIdx.x == 0) {
    const int leaf_index_ref = is_smaller ? smaller_leaf_index : larger_leaf_index;
    const unsigned buffer_write_pos = static_cast<unsigned int>(leaf_index_ref) + leaf_block_index * num_leaves;
    CUDASplitInfo* cuda_split_info = cuda_leaf_best_split_info + buffer_write_pos;
    const CUDASplitInfo* best_split_info = cuda_best_split_info + best_read_index;
    if (best_split_info->is_valid) {
      *cuda_split_info = *best_split_info;
      cuda_split_info->inner_feature_index = is_smaller ? cuda_task_feature_index[best_read_index] :
        cuda_task_feature_index[static_cast<int>(best_read_index) - num_tasks];
      cuda_split_info->is_valid = true;
    } else {
      cuda_split_info->gain = kMinScore;
      cuda_split_info->is_valid = false;
    }
  }
}

__global__ void SyncBestSplitForLeafKernelAllBlocks(
  const int smaller_leaf_index,
  const int larger_leaf_index,
  const unsigned int num_blocks_per_leaf,
  const int num_leaves,
  CUDASplitInfo* cuda_leaf_best_split_info,
  const bool larger_only) {
  if (!larger_only) {
    if (blockIdx.x == 0) {
      CUDASplitInfo* smaller_leaf_split_info = cuda_leaf_best_split_info + smaller_leaf_index;
      for (unsigned int block_index = 1; block_index < num_blocks_per_leaf; ++block_index) {
        const unsigned int leaf_read_pos = static_cast<unsigned int>(smaller_leaf_index) + block_index * static_cast<unsigned int>(num_leaves);
        const CUDASplitInfo* other_split_info = cuda_leaf_best_split_info + leaf_read_pos;
        if ((other_split_info->is_valid && smaller_leaf_split_info->is_valid &&
          other_split_info->gain > smaller_leaf_split_info->gain) ||
            (!smaller_leaf_split_info->is_valid && other_split_info->is_valid)) {
            smaller_leaf_split_info->is_valid = other_split_info->is_valid;
            smaller_leaf_split_info->inner_feature_index = other_split_info->inner_feature_index;
            smaller_leaf_split_info->default_left = other_split_info->default_left;
            smaller_leaf_split_info->threshold = other_split_info->threshold;
            smaller_leaf_split_info->gain = other_split_info->gain;
            smaller_leaf_split_info->left_sum_gradients = other_split_info->left_sum_gradients;
            smaller_leaf_split_info->left_sum_hessians = other_split_info->left_sum_hessians;
            smaller_leaf_split_info->left_count = other_split_info->left_count;
            smaller_leaf_split_info->left_gain = other_split_info->left_gain;
            smaller_leaf_split_info->left_value = other_split_info->left_value;
            smaller_leaf_split_info->right_sum_gradients = other_split_info->right_sum_gradients;
            smaller_leaf_split_info->right_sum_hessians = other_split_info->right_sum_hessians;
            smaller_leaf_split_info->right_count = other_split_info->right_count;
            smaller_leaf_split_info->right_gain = other_split_info->right_gain;
            smaller_leaf_split_info->right_value = other_split_info->right_value;
        }
      }
    }
  }
  if (larger_leaf_index >= 0) {
    if (blockIdx.x == 1 || larger_only) {
      CUDASplitInfo* larger_leaf_split_info = cuda_leaf_best_split_info + larger_leaf_index;
      for (unsigned int block_index = 1; block_index < num_blocks_per_leaf; ++block_index) {
        const unsigned int leaf_read_pos = static_cast<unsigned int>(larger_leaf_index) + block_index * static_cast<unsigned int>(num_leaves);
        const CUDASplitInfo* other_split_info = cuda_leaf_best_split_info + leaf_read_pos;
        if ((other_split_info->is_valid && larger_leaf_split_info->is_valid &&
          other_split_info->gain > larger_leaf_split_info->gain) ||
            (!larger_leaf_split_info->is_valid && other_split_info->is_valid)) {
            larger_leaf_split_info->is_valid = other_split_info->is_valid;
            larger_leaf_split_info->inner_feature_index = other_split_info->inner_feature_index;
            larger_leaf_split_info->default_left = other_split_info->default_left;
            larger_leaf_split_info->threshold = other_split_info->threshold;
            larger_leaf_split_info->gain = other_split_info->gain;
            larger_leaf_split_info->left_sum_gradients = other_split_info->left_sum_gradients;
            larger_leaf_split_info->left_sum_hessians = other_split_info->left_sum_hessians;
            larger_leaf_split_info->left_count = other_split_info->left_count;
            larger_leaf_split_info->left_gain = other_split_info->left_gain;
            larger_leaf_split_info->left_value = other_split_info->left_value;
            larger_leaf_split_info->right_sum_gradients = other_split_info->right_sum_gradients;
            larger_leaf_split_info->right_sum_hessians = other_split_info->right_sum_hessians;
            larger_leaf_split_info->right_count = other_split_info->right_count;
            larger_leaf_split_info->right_gain = other_split_info->right_gain;
            larger_leaf_split_info->right_value = other_split_info->right_value;
        }
      }
    }
  }
}

__global__ void SetInvalidLeafSplitInfoKernel(
  CUDASplitInfo* cuda_leaf_best_split_info,
  const bool is_smaller_leaf_valid,
  const bool is_larger_leaf_valid,
  const int smaller_leaf_index,
  const int larger_leaf_index) {
  if (!is_smaller_leaf_valid) {
    cuda_leaf_best_split_info[smaller_leaf_index].is_valid = false;
  }
  if (!is_larger_leaf_valid && larger_leaf_index >= 0) {
    cuda_leaf_best_split_info[larger_leaf_index].is_valid = false;
  }
}

void CUDABestSplitFinder::LaunchSyncBestSplitForLeafKernel(
  const int host_smaller_leaf_index,
  const int host_larger_leaf_index,
  const bool is_smaller_leaf_valid,
  const bool is_larger_leaf_valid) {
  if (!is_smaller_leaf_valid || !is_larger_leaf_valid) {
    SetInvalidLeafSplitInfoKernel<<<1, 1>>>(
      cuda_leaf_best_split_info_,
      is_smaller_leaf_valid, is_larger_leaf_valid,
      host_smaller_leaf_index, host_larger_leaf_index);
  }
  if (!is_smaller_leaf_valid && !is_larger_leaf_valid) {
    return;
  }
  int num_tasks = num_tasks_;
  int num_tasks_aligned = 1;
  num_tasks -= 1;
  while (num_tasks > 0) {
    num_tasks_aligned <<= 1;
    num_tasks >>= 1;
  }
  const int num_blocks_per_leaf = (num_tasks_ + NUM_TASKS_PER_SYNC_BLOCK - 1) / NUM_TASKS_PER_SYNC_BLOCK;
  if (host_larger_leaf_index >= 0 && is_smaller_leaf_valid && is_larger_leaf_valid) {
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK, 0, cuda_streams_[0]>>>(
      host_smaller_leaf_index,
      host_larger_leaf_index,
      cuda_leaf_best_split_info_,
      cuda_task_feature_index_,
      cuda_best_split_info_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      false,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1, 0, cuda_streams_[0]>>>(
        host_smaller_leaf_index,
        host_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_info_,
        false);
    }
    SynchronizeCUDADevice(__FILE__, __LINE__);
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK, 0, cuda_streams_[1]>>>(
      host_smaller_leaf_index,
      host_larger_leaf_index,
      cuda_leaf_best_split_info_,
      cuda_task_feature_index_,
      cuda_best_split_info_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      true,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1, 0, cuda_streams_[1]>>>(
        host_smaller_leaf_index,
        host_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_info_,
        true);
    }
  } else {
    const bool larger_only = (!is_smaller_leaf_valid && is_larger_leaf_valid);
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK>>>(
      host_smaller_leaf_index,
      host_larger_leaf_index,
      cuda_leaf_best_split_info_,
      cuda_task_feature_index_,
      cuda_best_split_info_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      larger_only,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SynchronizeCUDADevice(__FILE__, __LINE__);
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1>>>(
        host_smaller_leaf_index,
        host_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_info_,
        larger_only);
    }
  }
}

__global__ void FindBestFromAllSplitsKernel(const int cur_num_leaves,
  CUDASplitInfo* cuda_leaf_best_split_info,
  int* cuda_best_split_info_buffer) {
  __shared__ double gain_shared_buffer[32];
  __shared__ int leaf_index_shared_buffer[32];
  double thread_best_gain = kMinScore;
  int thread_best_leaf_index = -1;
  const int threadIdx_x = static_cast<int>(threadIdx.x);
  for (int leaf_index = threadIdx_x; leaf_index < cur_num_leaves; leaf_index += static_cast<int>(blockDim.x)) {
    const double leaf_best_gain = cuda_leaf_best_split_info[leaf_index].gain;
    if (cuda_leaf_best_split_info[leaf_index].is_valid && leaf_best_gain > thread_best_gain) {
      thread_best_gain = leaf_best_gain;
      thread_best_leaf_index = leaf_index;
    }
  }
  const int best_leaf_index = ReduceBestGainForLeaves(thread_best_gain, thread_best_leaf_index, gain_shared_buffer, leaf_index_shared_buffer);
  if (threadIdx_x == 0) {
    cuda_best_split_info_buffer[6] = best_leaf_index;
    if (best_leaf_index != -1) {
      cuda_leaf_best_split_info[best_leaf_index].is_valid = false;
      cuda_leaf_best_split_info[cur_num_leaves].is_valid = false;
      cuda_best_split_info_buffer[7] = cuda_leaf_best_split_info[best_leaf_index].num_cat_threshold;
    }
  }
}

__global__ void PrepareLeafBestSplitInfo(const int smaller_leaf_index, const int larger_leaf_index,
  int* cuda_best_split_info_buffer,
  const CUDASplitInfo* cuda_leaf_best_split_info) {
  const unsigned int threadIdx_x = blockIdx.x;
  if (threadIdx_x == 0) {
    cuda_best_split_info_buffer[0] = cuda_leaf_best_split_info[smaller_leaf_index].inner_feature_index;
  } else if (threadIdx_x == 1) {
    cuda_best_split_info_buffer[1] = cuda_leaf_best_split_info[smaller_leaf_index].threshold;
  } else if (threadIdx_x == 2) {
    cuda_best_split_info_buffer[2] = cuda_leaf_best_split_info[smaller_leaf_index].default_left;
  }
  if (larger_leaf_index >= 0) {
    if (threadIdx_x == 3) {
      cuda_best_split_info_buffer[3] = cuda_leaf_best_split_info[larger_leaf_index].inner_feature_index;
    } else if (threadIdx_x == 4) {
      cuda_best_split_info_buffer[4] = cuda_leaf_best_split_info[larger_leaf_index].threshold;
    } else if (threadIdx_x == 5) {
      cuda_best_split_info_buffer[5] = cuda_leaf_best_split_info[larger_leaf_index].default_left;
    }
  }
}

void CUDABestSplitFinder::LaunchFindBestFromAllSplitsKernel(
  const int cur_num_leaves,
  const int smaller_leaf_index, const int larger_leaf_index,
  int* smaller_leaf_best_split_feature,
  uint32_t* smaller_leaf_best_split_threshold,
  uint8_t* smaller_leaf_best_split_default_left,
  int* larger_leaf_best_split_feature,
  uint32_t* larger_leaf_best_split_threshold,
  uint8_t* larger_leaf_best_split_default_left,
  int* best_leaf_index,
  int* num_cat_threshold) {
  FindBestFromAllSplitsKernel<<<1, NUM_THREADS_FIND_BEST_LEAF, 0, cuda_streams_[1]>>>(cur_num_leaves,
    cuda_leaf_best_split_info_,
    cuda_best_split_info_buffer_);
  PrepareLeafBestSplitInfo<<<6, 1, 0, cuda_streams_[0]>>>(smaller_leaf_index, larger_leaf_index,
    cuda_best_split_info_buffer_,
    cuda_leaf_best_split_info_);
  std::vector<int> host_leaf_best_split_info_buffer(8, 0);
  SynchronizeCUDADevice(__FILE__, __LINE__);
  CopyFromCUDADeviceToHost<int>(host_leaf_best_split_info_buffer.data(), cuda_best_split_info_buffer_, 8, __FILE__, __LINE__);
  *smaller_leaf_best_split_feature = host_leaf_best_split_info_buffer[0];
  *smaller_leaf_best_split_threshold = static_cast<uint32_t>(host_leaf_best_split_info_buffer[1]);
  *smaller_leaf_best_split_default_left = static_cast<uint8_t>(host_leaf_best_split_info_buffer[2]);
  if (larger_leaf_index >= 0) {
    *larger_leaf_best_split_feature = host_leaf_best_split_info_buffer[3];
    *larger_leaf_best_split_threshold = static_cast<uint32_t>(host_leaf_best_split_info_buffer[4]);
    *larger_leaf_best_split_default_left = static_cast<uint8_t>(host_leaf_best_split_info_buffer[5]);
  }
  *best_leaf_index = host_leaf_best_split_info_buffer[6];
  *num_cat_threshold = host_leaf_best_split_info_buffer[7];
}

__global__ void AllocateCatVectorsKernel(
  CUDASplitInfo* cuda_split_infos, size_t len,
  const int max_num_categories_in_split,
  const bool has_categorical_feature) {
  const size_t i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < len) {
    if (has_categorical_feature) {
      cuda_split_infos[i].cat_threshold = new uint32_t[max_num_categories_in_split];
      cuda_split_infos[i].cat_threshold_real = new int[max_num_categories_in_split];
      cuda_split_infos[i].num_cat_threshold = 0;
    } else {
      cuda_split_infos[i].cat_threshold = nullptr;
      cuda_split_infos[i].cat_threshold_real = nullptr;
      cuda_split_infos[i].num_cat_threshold = 0;
    }
  }
}

void CUDABestSplitFinder::LaunchAllocateCatVectorsKernel(CUDASplitInfo* cuda_split_infos, size_t len) const {
  const int max_num_categories_in_split = min(max_cat_threshold_ / 2, max_num_categorical_bin_);
  const int num_blocks = (static_cast<int>(len) + NUM_THREADS_PER_BLOCK_BEST_SPLIT_FINDER - 1) / NUM_THREADS_PER_BLOCK_BEST_SPLIT_FINDER;
  AllocateCatVectorsKernel<<<num_blocks, NUM_THREADS_PER_BLOCK_BEST_SPLIT_FINDER>>>(
    cuda_split_infos, len, max_num_categories_in_split, has_categorical_feature_);
}

}  // namespace LightGBM

#endif  // USE_CUDA
