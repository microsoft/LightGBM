
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for license information.
 */

#ifdef USE_CUDA_EXP

#include <LightGBM/cuda/cuda_algorithms.hpp>

namespace LightGBM {

template <typename T>
__global__ void ShufflePrefixSumGlobalKernel(T* values, size_t len, T* block_prefix_sum_buffer) {
  __shared__ T shared_mem_buffer[32];
  const size_t index = static_cast<size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  T value = 0;
  if (index < len) {
    value = values[index];
  }
  const T prefix_sum_value = ShufflePrefixSum<T>(value, shared_mem_buffer);
  values[index] = prefix_sum_value;
  if (threadIdx.x == blockDim.x - 1) {
    block_prefix_sum_buffer[blockIdx.x] = prefix_sum_value;
  }
}

template <typename T>
__global__ void ShufflePrefixSumGlobalReduceBlockKernel(T* block_prefix_sum_buffer, int num_blocks) {
  __shared__ T shared_mem_buffer[32];
  const int num_blocks_per_thread = (num_blocks + GLOBAL_PREFIX_SUM_BLOCK_SIZE - 2) / (GLOBAL_PREFIX_SUM_BLOCK_SIZE - 1);
  int thread_block_start = threadIdx.x == 0 ? 0 : (threadIdx.x - 1) * num_blocks_per_thread;
  int thread_block_end = threadIdx.x == 0 ? 0 : min(thread_block_start + num_blocks_per_thread, num_blocks);
  T base = 0;
  for (int block_index = thread_block_start; block_index < thread_block_end; ++block_index) {
    base += block_prefix_sum_buffer[block_index];
  }
  base = ShufflePrefixSum<T>(base, shared_mem_buffer);
  thread_block_start = threadIdx.x == blockDim.x - 1 ? 0 : threadIdx.x * num_blocks_per_thread;
  thread_block_end = threadIdx.x == blockDim.x - 1 ? 0 : min(thread_block_start + num_blocks_per_thread, num_blocks);
  for (int block_index = thread_block_start + 1; block_index < thread_block_end; ++block_index) {
    block_prefix_sum_buffer[block_index] += block_prefix_sum_buffer[block_index - 1];
  }
  for (int block_index = thread_block_start; block_index < thread_block_end; ++block_index) {
    block_prefix_sum_buffer[block_index] += base;
  }
}

template <typename T>
__global__ void ShufflePrefixSumGlobalAddBase(size_t len, const T* block_prefix_sum_buffer, T* values) {
  const T base = blockIdx.x == 0 ? 0 : block_prefix_sum_buffer[blockIdx.x - 1];
  const size_t index = static_cast<size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (index < len) {
    values[index] += base;
  }
}

template <typename T>
void ShufflePrefixSumGlobalInner(T* values, size_t len, T* block_prefix_sum_buffer) {
  const int num_blocks = (static_cast<int>(len) + GLOBAL_PREFIX_SUM_BLOCK_SIZE - 1) / GLOBAL_PREFIX_SUM_BLOCK_SIZE;
  ShufflePrefixSumGlobalKernel<<<num_blocks, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(values, len, block_prefix_sum_buffer);
  ShufflePrefixSumGlobalReduceBlockKernel<<<1, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(block_prefix_sum_buffer, num_blocks);
  ShufflePrefixSumGlobalAddBase<<<num_blocks, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(len, block_prefix_sum_buffer, values);
}

template <>
void ShufflePrefixSumGlobal(uint16_t* values, size_t len, uint16_t* block_prefix_sum_buffer) {
  ShufflePrefixSumGlobalInner<uint16_t>(values, len, block_prefix_sum_buffer);
}

template <>
void ShufflePrefixSumGlobal(uint32_t* values, size_t len, uint32_t* block_prefix_sum_buffer) {
  ShufflePrefixSumGlobalInner<uint32_t>(values, len, block_prefix_sum_buffer);
}

template <>
void ShufflePrefixSumGlobal(uint64_t* values, size_t len, uint64_t* block_prefix_sum_buffer) {
  ShufflePrefixSumGlobalInner<uint64_t>(values, len, block_prefix_sum_buffer);
}

__global__ void BitonicArgSortItemsGlobalKernel(const double* scores,
  const int num_queries,
  const data_size_t* cuda_query_boundaries,
  data_size_t* out_indices) {
  const int query_index_start = static_cast<int>(blockIdx.x) * BITONIC_SORT_QUERY_ITEM_BLOCK_SIZE;
  const int query_index_end = min(query_index_start + BITONIC_SORT_QUERY_ITEM_BLOCK_SIZE, num_queries);
  for (int query_index = query_index_start; query_index < query_index_end; ++query_index) {
    const data_size_t query_item_start = cuda_query_boundaries[query_index];
    const data_size_t query_item_end = cuda_query_boundaries[query_index + 1];
    const data_size_t num_items_in_query = query_item_end - query_item_start;
    BitonicArgSortDevice<double, data_size_t, false, BITONIC_SORT_NUM_ELEMENTS, 11>(scores + query_item_start,
          out_indices + query_item_start,
          num_items_in_query);
    __syncthreads();
  }
}

void BitonicArgSortItemsGlobal(
  const double* scores,
  const int num_queries,
  const data_size_t* cuda_query_boundaries,
  data_size_t* out_indices) {
  const int num_blocks = (num_queries + BITONIC_SORT_QUERY_ITEM_BLOCK_SIZE - 1) / BITONIC_SORT_QUERY_ITEM_BLOCK_SIZE;
  BitonicArgSortItemsGlobalKernel<<<num_blocks, BITONIC_SORT_NUM_ELEMENTS>>>(
  scores, num_queries, cuda_query_boundaries, out_indices);
  SynchronizeCUDADevice(__FILE__, __LINE__);
}

}  // namespace LightGBM

#endif  // USE_CUDA_EXP
