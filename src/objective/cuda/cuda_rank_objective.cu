
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA_EXP

#include "cuda_rank_objective.hpp"

#include <LightGBM/cuda/cuda_algorithms.hpp>
#include <random>
#include <algorithm>

namespace LightGBM {

template <bool MAX_ITEM_GREATER_THAN_1024, data_size_t NUM_RANK_LABEL>
__global__ void GetGradientsKernel_LambdarankNDCG(const double* cuda_scores, const label_t* cuda_labels, const data_size_t num_data,
  const data_size_t num_queries, const data_size_t* cuda_query_boundaries, const double* cuda_inverse_max_dcgs,
  const bool norm, const double sigmoid, const int truncation_level, const double* cuda_label_gain, const data_size_t num_rank_label,
  score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  __shared__ score_t shared_scores[MAX_ITEM_GREATER_THAN_1024 ? 2048 : 1024];
  __shared__ uint16_t shared_indices[MAX_ITEM_GREATER_THAN_1024 ? 2048 : 1024];
  __shared__ score_t shared_lambdas[MAX_ITEM_GREATER_THAN_1024 ? 2048 : 1024];
  __shared__ score_t shared_hessians[MAX_ITEM_GREATER_THAN_1024 ? 2048 : 1024];
  __shared__ double shared_label_gain[NUM_RANK_LABEL > 1024 ? 1 : NUM_RANK_LABEL];
  const double* label_gain_ptr = nullptr;
  if (NUM_RANK_LABEL <= 1024) {
    for (uint32_t i = threadIdx.x; i < num_rank_label; i += blockDim.x) {
      shared_label_gain[i] = cuda_label_gain[i];
    }
    __syncthreads();
    label_gain_ptr = shared_label_gain;
  } else {
    label_gain_ptr = cuda_label_gain;
  }
  const data_size_t query_index_start = static_cast<data_size_t>(blockIdx.x) * NUM_QUERY_PER_BLOCK;
  const data_size_t query_index_end = min(query_index_start + NUM_QUERY_PER_BLOCK, num_queries);
  for (data_size_t query_index = query_index_start; query_index < query_index_end; ++query_index) {
    const double inverse_max_dcg = cuda_inverse_max_dcgs[query_index];
    const data_size_t query_start = cuda_query_boundaries[query_index];
    const data_size_t query_end = cuda_query_boundaries[query_index + 1];
    const data_size_t query_item_count = query_end - query_start;
    const double* cuda_scores_pointer = cuda_scores + query_start;
    score_t* cuda_out_gradients_pointer = cuda_out_gradients + query_start;
    score_t* cuda_out_hessians_pointer = cuda_out_hessians + query_start;
    const label_t* cuda_label_pointer = cuda_labels + query_start;
    if (threadIdx.x < query_item_count) {
      shared_scores[threadIdx.x] = cuda_scores_pointer[threadIdx.x];
      shared_indices[threadIdx.x] = static_cast<uint16_t>(threadIdx.x);
      shared_lambdas[threadIdx.x] = 0.0f;
      shared_hessians[threadIdx.x] = 0.0f;
    } else {
      shared_scores[threadIdx.x] = kMinScore;
      shared_indices[threadIdx.x] = static_cast<uint16_t>(threadIdx.x);
    }
    if (MAX_ITEM_GREATER_THAN_1024) {
      if (query_item_count > 1024) {
        const unsigned int threadIdx_x_plus_1024 = threadIdx.x + 1024;
        if (threadIdx_x_plus_1024 < query_item_count) {
          shared_scores[threadIdx_x_plus_1024] = cuda_scores_pointer[threadIdx_x_plus_1024];
          shared_indices[threadIdx_x_plus_1024] = static_cast<uint16_t>(threadIdx_x_plus_1024);
          shared_lambdas[threadIdx_x_plus_1024] = 0.0f;
          shared_hessians[threadIdx_x_plus_1024] = 0.0f;
        } else {
          shared_scores[threadIdx_x_plus_1024] = kMinScore;
          shared_indices[threadIdx_x_plus_1024] = static_cast<uint16_t>(threadIdx_x_plus_1024);
        }
      }
    }
    __syncthreads();
    if (MAX_ITEM_GREATER_THAN_1024) {
      if (query_item_count > 1024) {
        BitonicArgSort_2048<score_t, uint16_t, false>(shared_scores, shared_indices);
      } else {
        BitonicArgSort_1024<score_t, uint16_t, false>(shared_scores, shared_indices, static_cast<uint16_t>(query_item_count));
      }
    } else {
      BitonicArgSort_1024<score_t, uint16_t, false>(shared_scores, shared_indices, static_cast<uint16_t>(query_item_count));
    }
    __syncthreads();
    // get best and worst score
    const double best_score = shared_scores[shared_indices[0]];
    data_size_t worst_idx = query_item_count - 1;
    if (worst_idx > 0 && shared_scores[shared_indices[worst_idx]] == kMinScore) {
      worst_idx -= 1;
    }
    const double worst_score = shared_scores[shared_indices[worst_idx]];
    __shared__ double sum_lambdas;
    if (threadIdx.x == 0) {
      sum_lambdas = 0.0f;
    }
    __syncthreads();
    // start accumulate lambdas by pairs that contain at least one document above truncation level
    const data_size_t num_items_i = min(query_item_count - 1, truncation_level);
    const data_size_t num_j_per_i = query_item_count - 1;
    const data_size_t s = num_j_per_i - num_items_i + 1;
    const data_size_t num_pairs = (num_j_per_i + s) * num_items_i / 2;
    double thread_sum_lambdas = 0.0f;
    for (data_size_t pair_index = static_cast<data_size_t>(threadIdx.x); pair_index < num_pairs; pair_index += static_cast<data_size_t>(blockDim.x)) {
      const double square = 2 * static_cast<double>(pair_index) + s * s - s;
      const double sqrt_result = floor(sqrt(square));
      const data_size_t row_index = static_cast<data_size_t>(floor(sqrt(square - sqrt_result)) + 1 - s);
      const data_size_t i = num_items_i - 1 - row_index;
      const data_size_t j = num_j_per_i - (pair_index - (2 * s + row_index - 1) * row_index / 2);
      if (cuda_label_pointer[shared_indices[i]] != cuda_label_pointer[shared_indices[j]] && shared_scores[shared_indices[j]] != kMinScore) {
        data_size_t high_rank, low_rank;
        if (cuda_label_pointer[shared_indices[i]] > cuda_label_pointer[shared_indices[j]]) {
          high_rank = i;
          low_rank = j;
        } else {
          high_rank = j;
          low_rank = i;
        }
        const data_size_t high = shared_indices[high_rank];
        const int high_label = static_cast<int>(cuda_label_pointer[high]);
        const double high_score = shared_scores[high];
        const double high_label_gain = label_gain_ptr[high_label];
        const double high_discount = log2(2.0f + high_rank);
        const data_size_t low = shared_indices[low_rank];
        const int low_label = static_cast<int>(cuda_label_pointer[low]);
        const double low_score = shared_scores[low];
        const double low_label_gain = label_gain_ptr[low_label];
        const double low_discount = log2(2.0f + low_rank);

        const double delta_score = high_score - low_score;

        // get dcg gap
        const double dcg_gap = high_label_gain - low_label_gain;
        // get discount of this pair
        const double paired_discount = fabs(high_discount - low_discount);
        // get delta NDCG
        double delta_pair_NDCG = dcg_gap * paired_discount * inverse_max_dcg;
        // regular the delta_pair_NDCG by score distance
        if (norm && best_score != worst_score) {
          delta_pair_NDCG /= (0.01f + fabs(delta_score));
        }
        // calculate lambda for this pair
        double p_lambda = 1.0f / (1.0f + exp(sigmoid * delta_score));
        double p_hessian = p_lambda * (1.0f - p_lambda);
        // update
        p_lambda *= -sigmoid * delta_pair_NDCG;
        p_hessian *= sigmoid * sigmoid * delta_pair_NDCG;
        atomicAdd_block(shared_lambdas + low, -static_cast<score_t>(p_lambda));
        atomicAdd_block(shared_hessians + low, static_cast<score_t>(p_hessian));
        atomicAdd_block(shared_lambdas + high, static_cast<score_t>(p_lambda));
        atomicAdd_block(shared_hessians + high, static_cast<score_t>(p_hessian));
        // lambda is negative, so use minus to accumulate
        thread_sum_lambdas -= 2 * p_lambda;
      }
    }
    atomicAdd_block(&sum_lambdas, thread_sum_lambdas);
    __syncthreads();
    if (norm && sum_lambdas > 0) {
      const double norm_factor = log2(1 + sum_lambdas) / sum_lambdas;
      if (threadIdx.x < static_cast<unsigned int>(query_item_count)) {
        cuda_out_gradients_pointer[threadIdx.x] = static_cast<score_t>(shared_lambdas[threadIdx.x] * norm_factor);
        cuda_out_hessians_pointer[threadIdx.x] = static_cast<score_t>(shared_hessians[threadIdx.x] * norm_factor);
      }
      if (MAX_ITEM_GREATER_THAN_1024) {
        if (query_item_count > 1024) {
          const unsigned int threadIdx_x_plus_1024 = threadIdx.x + 1024;
          if (threadIdx_x_plus_1024 < static_cast<unsigned int>(query_item_count)) {
            cuda_out_gradients_pointer[threadIdx_x_plus_1024] = static_cast<score_t>(shared_lambdas[threadIdx_x_plus_1024] * norm_factor);
            cuda_out_hessians_pointer[threadIdx_x_plus_1024] = static_cast<score_t>(shared_hessians[threadIdx_x_plus_1024] * norm_factor);
          }
        }
      }
    } else {
      if (threadIdx.x < static_cast<unsigned int>(query_item_count)) {
        cuda_out_gradients_pointer[threadIdx.x] = static_cast<score_t>(shared_lambdas[threadIdx.x]);
        cuda_out_hessians_pointer[threadIdx.x] = static_cast<score_t>(shared_hessians[threadIdx.x]);
      }
      if (MAX_ITEM_GREATER_THAN_1024) {
        if (query_item_count > 1024) {
          const unsigned int threadIdx_x_plus_1024 = threadIdx.x + 1024;
          if (threadIdx_x_plus_1024 < static_cast<unsigned int>(query_item_count)) {
            cuda_out_gradients_pointer[threadIdx_x_plus_1024] = static_cast<score_t>(shared_lambdas[threadIdx_x_plus_1024]);
            cuda_out_hessians_pointer[threadIdx_x_plus_1024] = static_cast<score_t>(shared_hessians[threadIdx_x_plus_1024]);
          }
        }
      }
    }
    __syncthreads();
  }
}

template <data_size_t NUM_RANK_LABEL>
__global__ void GetGradientsKernel_LambdarankNDCG_Sorted(
  const double* cuda_scores, const int* cuda_item_indices_buffer, const label_t* cuda_labels, const data_size_t num_data,
  const data_size_t num_queries, const data_size_t* cuda_query_boundaries, const double* cuda_inverse_max_dcgs,
  const bool norm, const double sigmoid, const int truncation_level, const double* cuda_label_gain, const data_size_t num_rank_label,
  score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  __shared__ double shared_label_gain[NUM_RANK_LABEL > 1024 ? 1 : NUM_RANK_LABEL];
  const double* label_gain_ptr = nullptr;
  if (NUM_RANK_LABEL <= 1024) {
    for (uint32_t i = threadIdx.x; i < static_cast<uint32_t>(num_rank_label); i += blockDim.x) {
      shared_label_gain[i] = cuda_label_gain[i];
    }
    __syncthreads();
    label_gain_ptr = shared_label_gain;
  } else {
    label_gain_ptr = cuda_label_gain;
  }
  const data_size_t query_index_start = static_cast<data_size_t>(blockIdx.x) * NUM_QUERY_PER_BLOCK;
  const data_size_t query_index_end = min(query_index_start + NUM_QUERY_PER_BLOCK, num_queries);
  for (data_size_t query_index = query_index_start; query_index < query_index_end; ++query_index) {
    const double inverse_max_dcg = cuda_inverse_max_dcgs[query_index];
    const data_size_t query_start = cuda_query_boundaries[query_index];
    const data_size_t query_end = cuda_query_boundaries[query_index + 1];
    const data_size_t query_item_count = query_end - query_start;
    const double* cuda_scores_pointer = cuda_scores + query_start;
    const int* cuda_item_indices_buffer_pointer = cuda_item_indices_buffer + query_start;
    score_t* cuda_out_gradients_pointer = cuda_out_gradients + query_start;
    score_t* cuda_out_hessians_pointer = cuda_out_hessians + query_start;
    const label_t* cuda_label_pointer = cuda_labels + query_start;
    // get best and worst score
    const double best_score = cuda_scores_pointer[cuda_item_indices_buffer_pointer[0]];
    data_size_t worst_idx = query_item_count - 1;
    if (worst_idx > 0 && cuda_scores_pointer[cuda_item_indices_buffer_pointer[worst_idx]] == kMinScore) {
      worst_idx -= 1;
    }
    const double worst_score = cuda_scores_pointer[cuda_item_indices_buffer_pointer[worst_idx]];
    __shared__ double sum_lambdas;
    if (threadIdx.x == 0) {
      sum_lambdas = 0.0f;
    }
    for (int item_index = static_cast<int>(threadIdx.x); item_index < query_item_count; item_index += static_cast<int>(blockDim.x)) {
      cuda_out_gradients_pointer[item_index] = 0.0f;
      cuda_out_hessians_pointer[item_index] = 0.0f;
    }
    __syncthreads();
    // start accumulate lambdas by pairs that contain at least one document above truncation level
    const data_size_t num_items_i = min(query_item_count - 1, truncation_level);
    const data_size_t num_j_per_i = query_item_count - 1;
    const data_size_t s = num_j_per_i - num_items_i + 1;
    const data_size_t num_pairs = (num_j_per_i + s) * num_items_i / 2;
    double thread_sum_lambdas = 0.0f;
    for (data_size_t pair_index = static_cast<data_size_t>(threadIdx.x); pair_index < num_pairs; pair_index += static_cast<data_size_t>(blockDim.x)) {
      const double square = 2 * static_cast<double>(pair_index) + s * s - s;
      const double sqrt_result = floor(sqrt(square));
      const data_size_t row_index = static_cast<data_size_t>(floor(sqrt(square - sqrt_result)) + 1 - s);
      const data_size_t i = num_items_i - 1 - row_index;
      const data_size_t j = num_j_per_i - (pair_index - (2 * s + row_index - 1) * row_index / 2);
      if (j > i) {
        // skip pairs with the same labels
        if (cuda_label_pointer[cuda_item_indices_buffer_pointer[i]] != cuda_label_pointer[cuda_item_indices_buffer_pointer[j]] && cuda_scores_pointer[cuda_item_indices_buffer_pointer[j]] != kMinScore) {
          data_size_t high_rank, low_rank;
          if (cuda_label_pointer[cuda_item_indices_buffer_pointer[i]] > cuda_label_pointer[cuda_item_indices_buffer_pointer[j]]) {
            high_rank = i;
            low_rank = j;
          } else {
            high_rank = j;
            low_rank = i;
          }
          const data_size_t high = cuda_item_indices_buffer_pointer[high_rank];
          const int high_label = static_cast<int>(cuda_label_pointer[high]);
          const double high_score = cuda_scores_pointer[high];
          const double high_label_gain = label_gain_ptr[high_label];
          const double high_discount = log2(2.0f + high_rank);
          const data_size_t low = cuda_item_indices_buffer_pointer[low_rank];
          const int low_label = static_cast<int>(cuda_label_pointer[low]);
          const double low_score = cuda_scores_pointer[low];
          const double low_label_gain = label_gain_ptr[low_label];
          const double low_discount = log2(2.0f + low_rank);

          const double delta_score = high_score - low_score;

          // get dcg gap
          const double dcg_gap = high_label_gain - low_label_gain;
          // get discount of this pair
          const double paired_discount = fabs(high_discount - low_discount);
          // get delta NDCG
          double delta_pair_NDCG = dcg_gap * paired_discount * inverse_max_dcg;
          // regular the delta_pair_NDCG by score distance
          if (norm && best_score != worst_score) {
            delta_pair_NDCG /= (0.01f + fabs(delta_score));
          }
          // calculate lambda for this pair
          double p_lambda = 1.0f / (1.0f + exp(sigmoid * delta_score));
          double p_hessian = p_lambda * (1.0f - p_lambda);
          // update
          p_lambda *= -sigmoid * delta_pair_NDCG;
          p_hessian *= sigmoid * sigmoid * delta_pair_NDCG;
          atomicAdd_block(cuda_out_gradients_pointer + low, -static_cast<score_t>(p_lambda));
          atomicAdd_block(cuda_out_hessians_pointer + low, static_cast<score_t>(p_hessian));
          atomicAdd_block(cuda_out_gradients_pointer + high, static_cast<score_t>(p_lambda));
          atomicAdd_block(cuda_out_hessians_pointer + high, static_cast<score_t>(p_hessian));
          // lambda is negative, so use minus to accumulate
          thread_sum_lambdas -= 2 * p_lambda;
        }
      }
    }
    atomicAdd_block(&sum_lambdas, thread_sum_lambdas);
    __syncthreads();
    if (norm && sum_lambdas > 0) {
      const double norm_factor = log2(1 + sum_lambdas) / sum_lambdas;
      for (int item_index = static_cast<int>(threadIdx.x); item_index < query_item_count; item_index += static_cast<int>(blockDim.x)) {
        cuda_out_gradients_pointer[item_index] *= norm_factor;
        cuda_out_hessians_pointer[item_index] *= norm_factor;
      }
    }
    __syncthreads();
  }
}

void CUDALambdarankNDCG::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_queries_ + NUM_QUERY_PER_BLOCK - 1) / NUM_QUERY_PER_BLOCK;
  const data_size_t num_rank_label = static_cast<int>(label_gain_.size());

  #define GetGradientsKernel_LambdarankNDCG_ARGS \
    score, cuda_labels_, num_data_, \
    num_queries_, cuda_query_boundaries_, cuda_inverse_max_dcgs_.RawData(), \
    norm_, sigmoid_, truncation_level_, cuda_label_gain_.RawData(), num_rank_label, \
    gradients, hessians

  #define GetGradientsKernel_LambdarankNDCG_Sorted_ARGS \
    score, cuda_item_indices_buffer_.RawData(), cuda_labels_, num_data_, \
    num_queries_, cuda_query_boundaries_, cuda_inverse_max_dcgs_.RawData(), \
    norm_, sigmoid_, truncation_level_, cuda_label_gain_.RawData(), num_rank_label, \
    gradients, hessians

  if (max_items_in_query_aligned_ <= 1024) {
    if (num_rank_label <= 32) {
      GetGradientsKernel_LambdarankNDCG<false, 32><<<num_blocks, max_items_in_query_aligned_>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else if (num_rank_label <= 64) {
      GetGradientsKernel_LambdarankNDCG<false, 64><<<num_blocks, max_items_in_query_aligned_>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else if (num_rank_label <= 128) {
      GetGradientsKernel_LambdarankNDCG<false, 128><<<num_blocks, max_items_in_query_aligned_>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else if (num_rank_label <= 256) {
      GetGradientsKernel_LambdarankNDCG<false, 256><<<num_blocks, max_items_in_query_aligned_>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else if (num_rank_label <= 512) {
      GetGradientsKernel_LambdarankNDCG<false, 512><<<num_blocks, max_items_in_query_aligned_>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else if (num_rank_label <= 1024) {
      GetGradientsKernel_LambdarankNDCG<false, 1024><<<num_blocks, max_items_in_query_aligned_>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else {
      GetGradientsKernel_LambdarankNDCG<false, 2048><<<num_blocks, max_items_in_query_aligned_>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    }
  } else if (max_items_in_query_aligned_ <= 2048) {
    if (num_rank_label <= 32) {
      GetGradientsKernel_LambdarankNDCG<true, 32><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else if (num_rank_label <= 64) {
      GetGradientsKernel_LambdarankNDCG<true, 64><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else if (num_rank_label <= 128) {
      GetGradientsKernel_LambdarankNDCG<true, 128><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else if (num_rank_label <= 256) {
      GetGradientsKernel_LambdarankNDCG<true, 256><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else if (num_rank_label <= 512) {
      GetGradientsKernel_LambdarankNDCG<true, 512><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else if (num_rank_label <= 1024) {
      GetGradientsKernel_LambdarankNDCG<true, 1024><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    } else {
      GetGradientsKernel_LambdarankNDCG<true, 2048><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_ARGS);
    }
  } else {
    BitonicArgSortItemsGlobal(score, num_queries_, cuda_query_boundaries_, cuda_item_indices_buffer_.RawData());
    if (num_rank_label <= 32) {
      GetGradientsKernel_LambdarankNDCG_Sorted<32><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_Sorted_ARGS);
    } else if (num_rank_label <= 64) {
      GetGradientsKernel_LambdarankNDCG_Sorted<64><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_Sorted_ARGS);
    } else if (num_rank_label <= 128) {
      GetGradientsKernel_LambdarankNDCG_Sorted<128><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_Sorted_ARGS);
    } else if (num_rank_label <= 256) {
      GetGradientsKernel_LambdarankNDCG_Sorted<256><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_Sorted_ARGS);
    } else if (num_rank_label <= 512) {
      GetGradientsKernel_LambdarankNDCG_Sorted<512><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_Sorted_ARGS);
    } else if (num_rank_label <= 1024) {
      GetGradientsKernel_LambdarankNDCG_Sorted<1024><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_Sorted_ARGS);
    } else {
      GetGradientsKernel_LambdarankNDCG_Sorted<2048><<<num_blocks, 1024>>>(GetGradientsKernel_LambdarankNDCG_Sorted_ARGS);
    }
  }
  SynchronizeCUDADevice(__FILE__, __LINE__);

  #undef GetGradientsKernel_LambdarankNDCG_ARGS
  #undef GetGradientsKernel_LambdarankNDCG_Sorted_ARGS
}


__device__ __forceinline__ double CUDAPhi(const label_t l, double g) {
  return pow(2.0f, static_cast<double>(l)) - g;
}

template <size_t SHARED_MEMORY_SIZE>
__global__ void GetGradientsKernel_RankXENDCG_SharedMemory(
  const double* cuda_scores,
  const label_t* cuda_labels,
  const double* cuda_item_rands,
  const data_size_t num_data,
  const data_size_t num_queries,
  const data_size_t* cuda_query_boundaries,
  score_t* cuda_out_gradients,
  score_t* cuda_out_hessians) {
  const data_size_t query_index_start = static_cast<data_size_t>(blockIdx.x) * NUM_QUERY_PER_BLOCK;
  const data_size_t query_index_end = min(query_index_start + NUM_QUERY_PER_BLOCK, num_queries);
  for (data_size_t query_index = query_index_start; query_index < query_index_end; ++query_index) {
    const data_size_t item_index_start = cuda_query_boundaries[query_index];
    const data_size_t item_index_end = cuda_query_boundaries[query_index + 1];
    const data_size_t query_item_count = item_index_end - item_index_start;
    score_t* cuda_out_gradients_pointer = cuda_out_gradients + item_index_start;
    score_t* cuda_out_hessians_pointer = cuda_out_hessians + item_index_start;
    const label_t* cuda_labels_pointer = cuda_labels + item_index_start;
    const double* cuda_scores_pointer = cuda_scores + item_index_start;
    const double* cuda_item_rands_pointer = cuda_item_rands + item_index_start;
    const data_size_t block_reduce_size = query_item_count >= 1024 ? 1024 : query_item_count;
    __shared__ double shared_rho[SHARED_MEMORY_SIZE];
    // assert that warpSize == 32
    __shared__ double shared_buffer[32];
    __shared__ double shared_params[SHARED_MEMORY_SIZE];
    __shared__ score_t shared_lambdas[SHARED_MEMORY_SIZE];
    __shared__ double reduce_result;
    if (query_item_count <= 1) {
      for (data_size_t i = 0; i <= query_item_count; ++i) {
        cuda_out_gradients_pointer[i] = 0.0f;
        cuda_out_hessians_pointer[i] = 0.0f;
      }
      __syncthreads();
    } else {
      // compute softmax
      double thread_reduce_result = kMinScore;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        const double rho = cuda_scores_pointer[i];
        shared_rho[i] = rho;
        if (rho > thread_reduce_result) {
          thread_reduce_result = rho;
        }
      }
      __syncthreads();
      thread_reduce_result = ShuffleReduceMax<double>(thread_reduce_result, shared_buffer, block_reduce_size);
      if (threadIdx.x == 0) {
        reduce_result = thread_reduce_result;
      }
      __syncthreads();
      thread_reduce_result = 0.0f;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        const double exp_value = exp(shared_rho[i] - reduce_result);
        shared_rho[i] = exp_value;
        thread_reduce_result += exp_value;
      }
      thread_reduce_result = ShuffleReduceSum<double>(thread_reduce_result, shared_buffer, block_reduce_size);
      if (threadIdx.x == 0) {
        reduce_result = thread_reduce_result;
      }
      __syncthreads();
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        shared_rho[i] /= reduce_result;
      }
      __syncthreads();

      // compute params
      thread_reduce_result = 0.0f;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        const double param_value = CUDAPhi(cuda_labels_pointer[i], cuda_item_rands_pointer[i]);
        shared_params[i] = param_value;
        thread_reduce_result += param_value;
      }
      thread_reduce_result = ShuffleReduceSum<double>(thread_reduce_result, shared_buffer, block_reduce_size);
      if (threadIdx.x == 0) {
        reduce_result = thread_reduce_result;
        reduce_result = 1.0f / max(kEpsilon, reduce_result);
      }
      __syncthreads();
      const double inv_denominator = reduce_result;
      thread_reduce_result = 0.0f;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        const double term = -shared_params[i] * inv_denominator + shared_rho[i];
        shared_lambdas[i] = static_cast<score_t>(term);
        shared_params[i] = term / (1.0f - shared_rho[i]);
        thread_reduce_result += shared_params[i];
      }
      thread_reduce_result = ShuffleReduceSum<double>(thread_reduce_result, shared_buffer, block_reduce_size);
      if (threadIdx.x == 0) {
        reduce_result = thread_reduce_result;
      }
      __syncthreads();
      const double sum_l1 = reduce_result;
      thread_reduce_result = 0.0f;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        const double term = shared_rho[i] * (sum_l1 - shared_params[i]);
        shared_lambdas[i] += static_cast<score_t>(term);
        shared_params[i] = term / (1.0f - shared_rho[i]);
        thread_reduce_result += shared_params[i];
      }
      thread_reduce_result = ShuffleReduceSum<double>(thread_reduce_result, shared_buffer, block_reduce_size);
      if (threadIdx.x == 0) {
        reduce_result = thread_reduce_result;
      }
      __syncthreads();
      const double sum_l2 = reduce_result;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        shared_lambdas[i] += static_cast<score_t>(shared_rho[i] * (sum_l2 - shared_params[i]));
        cuda_out_hessians_pointer[i] = static_cast<score_t>(shared_rho[i] * (1.0f - shared_rho[i]));
      }
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        cuda_out_gradients_pointer[i] = shared_lambdas[i];
      }
      __syncthreads();
    }
  }
}

__global__ void GetGradientsKernel_RankXENDCG_GlobalMemory(
  const double* cuda_scores,
  const label_t* cuda_labels,
  const double* cuda_item_rands,
  const data_size_t num_data,
  const data_size_t num_queries,
  const data_size_t* cuda_query_boundaries,
  double* cuda_params_buffer,
  score_t* cuda_out_gradients,
  score_t* cuda_out_hessians) {
  const data_size_t query_index_start = static_cast<data_size_t>(blockIdx.x) * NUM_QUERY_PER_BLOCK;
  const data_size_t query_index_end = min(query_index_start + NUM_QUERY_PER_BLOCK, num_queries);
  for (data_size_t query_index = query_index_start; query_index < query_index_end; ++query_index) {
    const data_size_t item_index_start = cuda_query_boundaries[query_index];
    const data_size_t item_index_end = cuda_query_boundaries[query_index + 1];
    const data_size_t query_item_count = item_index_end - item_index_start;
    score_t* cuda_out_gradients_pointer = cuda_out_gradients + item_index_start;
    score_t* cuda_out_hessians_pointer = cuda_out_hessians + item_index_start;
    const label_t* cuda_labels_pointer = cuda_labels + item_index_start;
    const double* cuda_scores_pointer = cuda_scores + item_index_start;
    const double* cuda_item_rands_pointer = cuda_item_rands + item_index_start;
    double* cuda_params_buffer_pointer = cuda_params_buffer + item_index_start;
    const data_size_t block_reduce_size = query_item_count > 1024 ? 1024 : query_item_count;
    // assert that warpSize == 32, so we use buffer size 1024 / 32 = 32
    __shared__ double shared_buffer[32];
    __shared__ double reduce_result;
    if (query_item_count <= 1) {
      for (data_size_t i = 0; i <= query_item_count; ++i) {
        cuda_out_gradients_pointer[i] = 0.0f;
        cuda_out_hessians_pointer[i] = 0.0f;
      }
      __syncthreads();
    } else {
      // compute softmax
      double thread_reduce_result = kMinScore;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        const double rho = cuda_scores_pointer[i];
        if (rho > thread_reduce_result) {
          thread_reduce_result = rho;
        }
      }
      __syncthreads();
      thread_reduce_result = ShuffleReduceMax<double>(thread_reduce_result, shared_buffer, block_reduce_size);
      if (threadIdx.x == 0) {
        reduce_result = thread_reduce_result;
      }
      __syncthreads();
      thread_reduce_result = 0.0f;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        const double exp_value = exp(cuda_scores_pointer[i] - reduce_result);
        cuda_out_hessians_pointer[i] = exp_value;
        thread_reduce_result += exp_value;
      }
      thread_reduce_result = ShuffleReduceSum<double>(thread_reduce_result, shared_buffer, block_reduce_size);
      if (threadIdx.x == 0) {
        reduce_result = thread_reduce_result;
      }
      __syncthreads();
      // store probability into hessians
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        cuda_out_hessians_pointer[i] /= reduce_result;
      }
      __syncthreads();

      // compute params
      thread_reduce_result = 0.0f;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        const double param_value = CUDAPhi(cuda_labels_pointer[i], cuda_item_rands_pointer[i]);
        cuda_params_buffer_pointer[i] = param_value;
        thread_reduce_result += param_value;
      }
      thread_reduce_result = ShuffleReduceSum<double>(thread_reduce_result, shared_buffer, block_reduce_size);
      if (threadIdx.x == 0) {
        reduce_result = thread_reduce_result;
        reduce_result = 1.0f / max(kEpsilon, reduce_result);
      }
      __syncthreads();
      const double inv_denominator = reduce_result;
      thread_reduce_result = 0.0f;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        const double term = -cuda_params_buffer_pointer[i] * inv_denominator + cuda_out_hessians_pointer[i];
        cuda_out_gradients_pointer[i] = static_cast<score_t>(term);
        const double param = term / (1.0f - cuda_out_hessians_pointer[i]);
        cuda_params_buffer_pointer[i] = param;
        thread_reduce_result += param;
      }
      thread_reduce_result = ShuffleReduceSum<double>(thread_reduce_result, shared_buffer, block_reduce_size);
      if (threadIdx.x == 0) {
        reduce_result = thread_reduce_result;
      }
      __syncthreads();
      const double sum_l1 = reduce_result;
      thread_reduce_result = 0.0f;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        const double term = cuda_out_hessians_pointer[i] * (sum_l1 - cuda_params_buffer_pointer[i]);
        cuda_out_gradients_pointer[i] += static_cast<score_t>(term);
        const double param = term / (1.0f - cuda_out_hessians_pointer[i]);
        cuda_params_buffer_pointer[i] = param;
        thread_reduce_result += param;
      }
      thread_reduce_result = ShuffleReduceSum<double>(thread_reduce_result, shared_buffer, block_reduce_size);
      if (threadIdx.x == 0) {
        reduce_result = thread_reduce_result;
      }
      __syncthreads();
      const double sum_l2 = reduce_result;
      for (data_size_t i = static_cast<data_size_t>(threadIdx.x); i < query_item_count; i += static_cast<data_size_t>(blockDim.x)) {
        const double prob = cuda_out_hessians_pointer[i];
        cuda_out_gradients_pointer[i] += static_cast<score_t>(prob * (sum_l2 - cuda_params_buffer_pointer[i]));
        cuda_out_hessians_pointer[i] = static_cast<score_t>(prob * (1.0f - prob));
      }
      __syncthreads();
    }
  }
}

void CUDARankXENDCG::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_queries_ + NUM_QUERY_PER_BLOCK - 1) / NUM_QUERY_PER_BLOCK;
  if (max_items_in_query_aligned_ <= 1024) {
    GetGradientsKernel_RankXENDCG_SharedMemory<1024><<<num_blocks, max_items_in_query_aligned_>>>(
      score,
      cuda_labels_,
      cuda_item_rands_,
      num_data_,
      num_queries_,
      cuda_query_boundaries_,
      gradients,
      hessians);
  } else if (max_items_in_query_aligned_ <= 2 * 1024) {
    GetGradientsKernel_RankXENDCG_SharedMemory<2 * 1024><<<num_blocks, 1024>>>(
      score,
      cuda_labels_,
      cuda_item_rands_,
      num_data_,
      num_queries_,
      cuda_query_boundaries_,
      gradients,
      hessians);
  } else {
    GetGradientsKernel_RankXENDCG_GlobalMemory<<<num_blocks, 1024>>>(
      score,
      cuda_labels_,
      cuda_item_rands_,
      num_data_,
      num_queries_,
      cuda_query_boundaries_,
      cuda_params_buffer_,
      gradients,
      hessians);
  }
  SynchronizeCUDADevice(__FILE__, __LINE__);
}


}  // namespace LightGBM

#endif  // USE_CUDA_EXP
