
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA_EXP

#include "cuda_histogram_constructor.hpp"

#include <LightGBM/cuda/cuda_algorithms.hpp>

#include <algorithm>

namespace LightGBM {

template <typename BIN_TYPE, typename HIST_TYPE, size_t SHARED_HIST_SIZE>
__global__ void CUDAConstructHistogramDenseKernel(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const uint32_t* column_hist_offsets,
  const uint32_t* column_hist_offsets_full,
  const int* feature_partition_column_index_offsets,
  const data_size_t num_data) {
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf = smaller_leaf_splits->num_data_in_leaf;
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = smaller_leaf_splits->data_indices_in_leaf;
  __shared__ HIST_TYPE shared_hist[SHARED_HIST_SIZE];
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const int partition_column_start = feature_partition_column_index_offsets[blockIdx.x];
  const int partition_column_end = feature_partition_column_index_offsets[blockIdx.x + 1];
  const BIN_TYPE* data_ptr = data + static_cast<size_t>(partition_column_start) * num_data;
  const int num_columns_in_partition = partition_column_end - partition_column_start;
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (static_cast<size_t>(blockIdx_y) * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const int column_index = static_cast<int>(threadIdx.x) + partition_column_start;
  if (threadIdx.x < static_cast<unsigned int>(num_columns_in_partition)) {
    HIST_TYPE* shared_hist_ptr = shared_hist + (column_hist_offsets[column_index] << 1);
    for (data_size_t inner_data_index = static_cast<data_size_t>(threadIdx.y); inner_data_index < block_num_data; inner_data_index += blockDim.y) {
      const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[static_cast<size_t>(data_index) * num_columns_in_partition + threadIdx.x]);
      const uint32_t pos = bin << 1;
      HIST_TYPE* pos_ptr = shared_hist_ptr + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
    }
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = smaller_leaf_splits->hist_in_leaf + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

template <typename BIN_TYPE, typename DATA_PTR_TYPE, typename HIST_TYPE, size_t SHARED_HIST_SIZE>
__global__ void CUDAConstructHistogramSparseKernel(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const DATA_PTR_TYPE* row_ptr,
  const DATA_PTR_TYPE* partition_ptr,
  const uint32_t* column_hist_offsets_full,
  const data_size_t num_data) {
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf = smaller_leaf_splits->num_data_in_leaf;
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = smaller_leaf_splits->data_indices_in_leaf;
  __shared__ HIST_TYPE shared_hist[SHARED_HIST_SIZE];
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const DATA_PTR_TYPE* block_row_ptr = row_ptr + static_cast<size_t>(blockIdx.x) * (num_data + 1);
  const BIN_TYPE* data_ptr = data + partition_ptr[blockIdx.x];
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  for (data_size_t i = 0; i < num_iteration_this; ++i) {
    const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
    const DATA_PTR_TYPE row_start = block_row_ptr[data_index];
    const DATA_PTR_TYPE row_end = block_row_ptr[data_index + 1];
    const DATA_PTR_TYPE row_size = row_end - row_start;
    if (threadIdx.x < row_size) {
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[row_start + threadIdx.x]);
      const uint32_t pos = bin << 1;
      HIST_TYPE* pos_ptr = shared_hist + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
    }
    inner_data_index += blockDim.y;
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = smaller_leaf_splits->hist_in_leaf + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

template <typename BIN_TYPE>
__global__ void CUDAConstructHistogramDenseKernel_GlobalMemory(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const uint32_t* column_hist_offsets,
  const uint32_t* column_hist_offsets_full,
  const int* feature_partition_column_index_offsets,
  const data_size_t num_data,
  float* global_hist_buffer) {
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf = smaller_leaf_splits->num_data_in_leaf;
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = smaller_leaf_splits->data_indices_in_leaf;
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const int partition_column_start = feature_partition_column_index_offsets[blockIdx.x];
  const int partition_column_end = feature_partition_column_index_offsets[blockIdx.x + 1];
  const BIN_TYPE* data_ptr = data + static_cast<size_t>(partition_column_start) * num_data;
  const int num_columns_in_partition = partition_column_end - partition_column_start;
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  const int num_total_bin = column_hist_offsets_full[gridDim.x];
  float* shared_hist = global_hist_buffer + (blockIdx.y * num_total_bin + partition_hist_start) * 2;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (static_cast<size_t>(blockIdx_y) * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  const int column_index = static_cast<int>(threadIdx.x) + partition_column_start;
  if (threadIdx.x < static_cast<unsigned int>(num_columns_in_partition)) {
    float* shared_hist_ptr = shared_hist + (column_hist_offsets[column_index] << 1);
    for (data_size_t i = 0; i < num_iteration_this; ++i) {
      const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[static_cast<size_t>(data_index) * num_columns_in_partition + threadIdx.x]);
      const uint32_t pos = bin << 1;
      float* pos_ptr = shared_hist_ptr + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
      inner_data_index += blockDim.y;
    }
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = smaller_leaf_splits->hist_in_leaf + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

template <typename BIN_TYPE, typename DATA_PTR_TYPE>
__global__ void CUDAConstructHistogramSparseKernel_GlobalMemory(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const score_t* cuda_gradients,
  const score_t* cuda_hessians,
  const BIN_TYPE* data,
  const DATA_PTR_TYPE* row_ptr,
  const DATA_PTR_TYPE* partition_ptr,
  const uint32_t* column_hist_offsets_full,
  const data_size_t num_data,
  float* global_hist_buffer) {
  const int dim_y = static_cast<int>(gridDim.y * blockDim.y);
  const data_size_t num_data_in_smaller_leaf = smaller_leaf_splits->num_data_in_leaf;
  const data_size_t num_data_per_thread = (num_data_in_smaller_leaf + dim_y - 1) / dim_y;
  const data_size_t* data_indices_ref = smaller_leaf_splits->data_indices_in_leaf;
  const unsigned int num_threads_per_block = blockDim.x * blockDim.y;
  const DATA_PTR_TYPE* block_row_ptr = row_ptr + static_cast<size_t>(blockIdx.x) * (num_data + 1);
  const BIN_TYPE* data_ptr = data + partition_ptr[blockIdx.x];
  const uint32_t partition_hist_start = column_hist_offsets_full[blockIdx.x];
  const uint32_t partition_hist_end = column_hist_offsets_full[blockIdx.x + 1];
  const uint32_t num_items_in_partition = (partition_hist_end - partition_hist_start) << 1;
  const unsigned int thread_idx = threadIdx.x + threadIdx.y * blockDim.x;
  const int num_total_bin = column_hist_offsets_full[gridDim.x];
  float* shared_hist = global_hist_buffer + (blockIdx.y * num_total_bin + partition_hist_start) * 2;
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    shared_hist[i] = 0.0f;
  }
  __syncthreads();
  const unsigned int threadIdx_y = threadIdx.y;
  const unsigned int blockIdx_y = blockIdx.y;
  const data_size_t block_start = (blockIdx_y * blockDim.y) * num_data_per_thread;
  const data_size_t* data_indices_ref_this_block = data_indices_ref + block_start;
  data_size_t block_num_data = max(0, min(num_data_in_smaller_leaf - block_start, num_data_per_thread * static_cast<data_size_t>(blockDim.y)));
  const data_size_t num_iteration_total = (block_num_data + blockDim.y - 1) / blockDim.y;
  const data_size_t remainder = block_num_data % blockDim.y;
  const data_size_t num_iteration_this = remainder == 0 ? num_iteration_total : num_iteration_total - static_cast<data_size_t>(threadIdx_y >= remainder);
  data_size_t inner_data_index = static_cast<data_size_t>(threadIdx_y);
  for (data_size_t i = 0; i < num_iteration_this; ++i) {
    const data_size_t data_index = data_indices_ref_this_block[inner_data_index];
    const DATA_PTR_TYPE row_start = block_row_ptr[data_index];
    const DATA_PTR_TYPE row_end = block_row_ptr[data_index + 1];
    const DATA_PTR_TYPE row_size = row_end - row_start;
    if (threadIdx.x < row_size) {
      const score_t grad = cuda_gradients[data_index];
      const score_t hess = cuda_hessians[data_index];
      const uint32_t bin = static_cast<uint32_t>(data_ptr[row_start + threadIdx.x]);
      const uint32_t pos = bin << 1;
      float* pos_ptr = shared_hist + pos;
      atomicAdd_block(pos_ptr, grad);
      atomicAdd_block(pos_ptr + 1, hess);
    }
    inner_data_index += blockDim.y;
  }
  __syncthreads();
  hist_t* feature_histogram_ptr = smaller_leaf_splits->hist_in_leaf + (partition_hist_start << 1);
  for (unsigned int i = thread_idx; i < num_items_in_partition; i += num_threads_per_block) {
    atomicAdd_system(feature_histogram_ptr + i, shared_hist[i]);
  }
}

void CUDAHistogramConstructor::LaunchConstructHistogramKernel(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const data_size_t num_data_in_smaller_leaf) {
  if (cuda_row_data_->shared_hist_size() == DP_SHARED_HIST_SIZE && gpu_use_dp_) {
    LaunchConstructHistogramKernelInner<double, DP_SHARED_HIST_SIZE>(cuda_smaller_leaf_splits, num_data_in_smaller_leaf);
  } else if (cuda_row_data_->shared_hist_size() == SP_SHARED_HIST_SIZE && !gpu_use_dp_) {
    LaunchConstructHistogramKernelInner<float, SP_SHARED_HIST_SIZE>(cuda_smaller_leaf_splits, num_data_in_smaller_leaf);
  } else {
    Log::Fatal("Unknown shared histogram size %d", cuda_row_data_->shared_hist_size());
  }
}

template <typename HIST_TYPE, size_t SHARED_HIST_SIZE>
void CUDAHistogramConstructor::LaunchConstructHistogramKernelInner(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const data_size_t num_data_in_smaller_leaf) {
  if (cuda_row_data_->bit_type() == 8) {
    LaunchConstructHistogramKernelInner0<HIST_TYPE, SHARED_HIST_SIZE, uint8_t>(cuda_smaller_leaf_splits, num_data_in_smaller_leaf);
  } else if (cuda_row_data_->bit_type() == 16) {
    LaunchConstructHistogramKernelInner0<HIST_TYPE, SHARED_HIST_SIZE, uint16_t>(cuda_smaller_leaf_splits, num_data_in_smaller_leaf);
  } else if (cuda_row_data_->bit_type() == 32) {
    LaunchConstructHistogramKernelInner0<HIST_TYPE, SHARED_HIST_SIZE, uint32_t>(cuda_smaller_leaf_splits, num_data_in_smaller_leaf);
  } else {
    Log::Fatal("Unknown bit_type = %d", cuda_row_data_->bit_type());
  }
}

template <typename HIST_TYPE, size_t SHARED_HIST_SIZE, typename BIN_TYPE>
void CUDAHistogramConstructor::LaunchConstructHistogramKernelInner0(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const data_size_t num_data_in_smaller_leaf) {
  if (cuda_row_data_->row_ptr_bit_type() == 16) {
    LaunchConstructHistogramKernelInner1<HIST_TYPE, SHARED_HIST_SIZE, BIN_TYPE, uint16_t>(cuda_smaller_leaf_splits, num_data_in_smaller_leaf);
  } else if (cuda_row_data_->row_ptr_bit_type() == 32) {
    LaunchConstructHistogramKernelInner1<HIST_TYPE, SHARED_HIST_SIZE, BIN_TYPE, uint32_t>(cuda_smaller_leaf_splits, num_data_in_smaller_leaf);
  } else if (cuda_row_data_->row_ptr_bit_type() == 64) {
    LaunchConstructHistogramKernelInner1<HIST_TYPE, SHARED_HIST_SIZE, BIN_TYPE, uint64_t>(cuda_smaller_leaf_splits, num_data_in_smaller_leaf);
  } else {
    if (!cuda_row_data_->is_sparse()) {
      LaunchConstructHistogramKernelInner1<HIST_TYPE, SHARED_HIST_SIZE, BIN_TYPE, uint16_t>(cuda_smaller_leaf_splits, num_data_in_smaller_leaf);
    } else {
      Log::Fatal("Unknown row_ptr_bit_type = %d", cuda_row_data_->row_ptr_bit_type());
    }
  }
}

template <typename HIST_TYPE, size_t SHARED_HIST_SIZE, typename BIN_TYPE, typename PTR_TYPE>
void CUDAHistogramConstructor::LaunchConstructHistogramKernelInner1(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const data_size_t num_data_in_smaller_leaf) {
  if (cuda_row_data_->NumLargeBinPartition() == 0) {
    LaunchConstructHistogramKernelInner2<HIST_TYPE, SHARED_HIST_SIZE, BIN_TYPE, PTR_TYPE, false>(cuda_smaller_leaf_splits, num_data_in_smaller_leaf);
  } else {
    LaunchConstructHistogramKernelInner2<HIST_TYPE, SHARED_HIST_SIZE, BIN_TYPE, PTR_TYPE, true>(cuda_smaller_leaf_splits, num_data_in_smaller_leaf);
  }
}

template <typename HIST_TYPE, size_t SHARED_HIST_SIZE, typename BIN_TYPE, typename PTR_TYPE, bool USE_GLOBAL_MEM_BUFFER>
void CUDAHistogramConstructor::LaunchConstructHistogramKernelInner2(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const data_size_t num_data_in_smaller_leaf) {
  int grid_dim_x = 0;
  int grid_dim_y = 0;
  int block_dim_x = 0;
  int block_dim_y = 0;
  CalcConstructHistogramKernelDim(&grid_dim_x, &grid_dim_y, &block_dim_x, &block_dim_y, num_data_in_smaller_leaf);
  dim3 grid_dim(grid_dim_x, grid_dim_y);
  dim3 block_dim(block_dim_x, block_dim_y);
  if (!USE_GLOBAL_MEM_BUFFER) {
    if (cuda_row_data_->is_sparse()) {
      CUDAConstructHistogramSparseKernel<BIN_TYPE, PTR_TYPE, HIST_TYPE, SHARED_HIST_SIZE><<<grid_dim, block_dim, 0, cuda_stream_>>>(
        cuda_smaller_leaf_splits,
        cuda_gradients_, cuda_hessians_,
        cuda_row_data_->GetBin<BIN_TYPE>(),
        cuda_row_data_->GetRowPtr<PTR_TYPE>(),
        cuda_row_data_->GetPartitionPtr<PTR_TYPE>(),
        cuda_row_data_->cuda_partition_hist_offsets(),
        num_data_);
    } else {
      CUDAConstructHistogramDenseKernel<BIN_TYPE, HIST_TYPE, SHARED_HIST_SIZE><<<grid_dim, block_dim, 0, cuda_stream_>>>(
        cuda_smaller_leaf_splits,
        cuda_gradients_, cuda_hessians_,
        cuda_row_data_->GetBin<BIN_TYPE>(),
        cuda_row_data_->cuda_column_hist_offsets(),
        cuda_row_data_->cuda_partition_hist_offsets(),
        cuda_row_data_->cuda_feature_partition_column_index_offsets(),
        num_data_);
    }
  } else {
    if (cuda_row_data_->is_sparse()) {
      CUDAConstructHistogramSparseKernel_GlobalMemory<BIN_TYPE, PTR_TYPE><<<grid_dim, block_dim, 0, cuda_stream_>>>(
        cuda_smaller_leaf_splits,
        cuda_gradients_, cuda_hessians_,
        cuda_row_data_->GetBin<BIN_TYPE>(),
        cuda_row_data_->GetRowPtr<PTR_TYPE>(),
        cuda_row_data_->GetPartitionPtr<PTR_TYPE>(),
        cuda_row_data_->cuda_partition_hist_offsets(),
        num_data_,
        cuda_hist_buffer_);
    } else {
      CUDAConstructHistogramDenseKernel_GlobalMemory<BIN_TYPE><<<grid_dim, block_dim, 0, cuda_stream_>>>(
        cuda_smaller_leaf_splits,
        cuda_gradients_, cuda_hessians_,
        cuda_row_data_->GetBin<BIN_TYPE>(),
        cuda_row_data_->cuda_column_hist_offsets(),
        cuda_row_data_->cuda_partition_hist_offsets(),
        cuda_row_data_->cuda_feature_partition_column_index_offsets(),
        num_data_,
        cuda_hist_buffer_);
    }
  }
}

__global__ void SubtractHistogramKernel(
  const int num_total_bin,
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const CUDALeafSplitsStruct* cuda_larger_leaf_splits) {
  const unsigned int global_thread_index = threadIdx.x + blockIdx.x * blockDim.x;
  const int cuda_larger_leaf_index = cuda_larger_leaf_splits->leaf_index;
  if (cuda_larger_leaf_index >= 0) {
    const hist_t* smaller_leaf_hist = cuda_smaller_leaf_splits->hist_in_leaf;
    hist_t* larger_leaf_hist = cuda_larger_leaf_splits->hist_in_leaf;
    if (global_thread_index < 2 * num_total_bin) {
      larger_leaf_hist[global_thread_index] -= smaller_leaf_hist[global_thread_index];
    }
  }
}

__global__ void FixHistogramKernel(
  const uint32_t* cuda_feature_num_bins,
  const uint32_t* cuda_feature_hist_offsets,
  const uint32_t* cuda_feature_most_freq_bins,
  const int* cuda_need_fix_histogram_features,
  const uint32_t* cuda_need_fix_histogram_features_num_bin_aligned,
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits) {
  __shared__ hist_t shared_mem_buffer[32];
  const unsigned int blockIdx_x = blockIdx.x;
  const int feature_index = cuda_need_fix_histogram_features[blockIdx_x];
  const uint32_t num_bin_aligned = cuda_need_fix_histogram_features_num_bin_aligned[blockIdx_x];
  const uint32_t feature_hist_offset = cuda_feature_hist_offsets[feature_index];
  const uint32_t most_freq_bin = cuda_feature_most_freq_bins[feature_index];
  const double leaf_sum_gradients = cuda_smaller_leaf_splits->sum_of_gradients;
  const double leaf_sum_hessians = cuda_smaller_leaf_splits->sum_of_hessians;
  hist_t* feature_hist = cuda_smaller_leaf_splits->hist_in_leaf + feature_hist_offset * 2;
  const unsigned int threadIdx_x = threadIdx.x;
  const uint32_t num_bin = cuda_feature_num_bins[feature_index];
  const uint32_t hist_pos = threadIdx_x << 1;
  const hist_t bin_gradient = (threadIdx_x < num_bin && threadIdx_x != most_freq_bin) ? feature_hist[hist_pos] : 0.0f;
  const hist_t bin_hessian = (threadIdx_x < num_bin && threadIdx_x != most_freq_bin) ? feature_hist[hist_pos + 1] : 0.0f;
  const hist_t sum_gradient = ShuffleReduceSum<hist_t>(bin_gradient, shared_mem_buffer, num_bin_aligned);
  const hist_t sum_hessian = ShuffleReduceSum<hist_t>(bin_hessian, shared_mem_buffer, num_bin_aligned);
  if (threadIdx_x == 0) {
    feature_hist[most_freq_bin << 1] = leaf_sum_gradients - sum_gradient;
    feature_hist[(most_freq_bin << 1) + 1] = leaf_sum_hessians - sum_hessian;
  }
}

void CUDAHistogramConstructor::LaunchSubtractHistogramKernel(
  const CUDALeafSplitsStruct* cuda_smaller_leaf_splits,
  const CUDALeafSplitsStruct* cuda_larger_leaf_splits) {
  const int num_subtract_threads = 2 * num_total_bin_;
  const int num_subtract_blocks = (num_subtract_threads + SUBTRACT_BLOCK_SIZE - 1) / SUBTRACT_BLOCK_SIZE;
  global_timer.Start("CUDAHistogramConstructor::FixHistogramKernel");
  if (need_fix_histogram_features_.size() > 0) {
    FixHistogramKernel<<<need_fix_histogram_features_.size(), FIX_HISTOGRAM_BLOCK_SIZE, 0, cuda_stream_>>>(
      cuda_feature_num_bins_,
      cuda_feature_hist_offsets_,
      cuda_feature_most_freq_bins_,
      cuda_need_fix_histogram_features_,
      cuda_need_fix_histogram_features_num_bin_aligned_,
      cuda_smaller_leaf_splits);
  }
  global_timer.Stop("CUDAHistogramConstructor::FixHistogramKernel");
  global_timer.Start("CUDAHistogramConstructor::SubtractHistogramKernel");
  SubtractHistogramKernel<<<num_subtract_blocks, SUBTRACT_BLOCK_SIZE, 0, cuda_stream_>>>(
    num_total_bin_,
    cuda_smaller_leaf_splits,
    cuda_larger_leaf_splits);
  global_timer.Stop("CUDAHistogramConstructor::SubtractHistogramKernel");
}

}  // namespace LightGBM

#endif  // USE_CUDA_EXP
