
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA_EXP

#include "cuda_regression_objective.hpp"
#include <LightGBM/cuda/cuda_algorithms.hpp>

namespace LightGBM {

double CUDARegressionL2loss::LaunchCalcInitScoreKernel() const {
  double label_sum = 0.0f, weight_sum = 0.0f;
  if (cuda_weights_ == nullptr) {
    ShuffleReduceSumGlobal<label_t, double>(cuda_labels_, static_cast<size_t>(num_data_), cuda_block_buffer_.RawData());
    CopyFromCUDADeviceToHost<double>(&label_sum, cuda_block_buffer_.RawData(), 1, __FILE__, __LINE__);
    weight_sum = static_cast<double>(num_data_);
  } else {
    ShuffleReduceDotProdGlobal<label_t, double>(cuda_labels_, cuda_weights_, static_cast<size_t>(num_data_), cuda_block_buffer_.RawData());
    CopyFromCUDADeviceToHost<double>(&label_sum, cuda_block_buffer_.RawData(), 1, __FILE__, __LINE__);
    ShuffleReduceSumGlobal<label_t, double>(cuda_weights_, static_cast<size_t>(num_data_), cuda_block_buffer_.RawData());
    CopyFromCUDADeviceToHost<double>(&weight_sum, cuda_block_buffer_.RawData(), 1, __FILE__, __LINE__);
  }
  return label_sum / weight_sum;
}

__global__ void ConvertOutputCUDAKernel_Regression(const bool sqrt, const data_size_t num_data, const double* input, double* output) {
  const int data_index = static_cast<data_size_t>(blockIdx.x * blockDim.x + threadIdx.x);
  if (data_index < num_data) {
    if (sqrt) {
      const double sign = input[data_index] >= 0.0f ? 1 : -1;
      output[data_index] = sign * input[data_index] * input[data_index];
    } else {
      output[data_index] = input[data_index];
    }
  }
}

void CUDARegressionL2loss::LaunchConvertOutputCUDAKernel(const data_size_t num_data, const double* input, double* output) const {
  const int num_blocks = (num_data + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  ConvertOutputCUDAKernel_Regression<<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(sqrt_, num_data, input, output);
}

template <bool USE_WEIGHT>
__global__ void GetGradientsKernel_RegressionL2(const double* cuda_scores, const label_t* cuda_labels, const label_t* cuda_weights, const data_size_t num_data,
  score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  const data_size_t data_index = static_cast<data_size_t>(blockDim.x * blockIdx.x + threadIdx.x);
  if (data_index < num_data) {
    if (!USE_WEIGHT) {
      cuda_out_gradients[data_index] = static_cast<score_t>(cuda_scores[data_index] - cuda_labels[data_index]);
      cuda_out_hessians[data_index] = 1.0f;
    } else {
      const score_t weight = static_cast<score_t>(cuda_weights[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>(cuda_scores[data_index] - cuda_labels[data_index]) * weight;
      cuda_out_hessians[data_index] = weight;
    }
  }
}

void CUDARegressionL2loss::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  if (cuda_weights_ == nullptr) {
    GetGradientsKernel_RegressionL2<false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, nullptr, num_data_, gradients, hessians);
  } else {
    GetGradientsKernel_RegressionL2<true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, cuda_weights_, num_data_, gradients, hessians);
  }
}


double CUDARegressionL1loss::LaunchCalcInitScoreKernel() const {
  const double alpha = 0.5f;
  if (cuda_weights_ == nullptr) {
    PercentileGlobal<label_t, data_size_t, label_t, double, false, false>(
      cuda_labels_, nullptr, cuda_data_indices_buffer_.RawData(), nullptr, nullptr, alpha, num_data_, cuda_percentile_result_.RawData());
  } else {
    PercentileGlobal<label_t, data_size_t, label_t, double, false, true>(
      cuda_labels_, cuda_weights_, cuda_data_indices_buffer_.RawData(), cuda_weights_prefix_sum_.RawData(),
      cuda_weights_prefix_sum_buffer_.RawData(), alpha, num_data_, cuda_percentile_result_.RawData());
  }
  label_t percentile_result = 0.0f;
  CopyFromCUDADeviceToHost<label_t>(&percentile_result, cuda_percentile_result_.RawData(), 1, __FILE__, __LINE__);
  SynchronizeCUDADevice(__FILE__, __LINE__);
  return static_cast<label_t>(percentile_result);
}

template <bool USE_WEIGHT>
__global__ void GetGradientsKernel_RegressionL1(const double* cuda_scores, const label_t* cuda_labels, const label_t* cuda_weights, const data_size_t num_data,
  score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  const data_size_t data_index = static_cast<data_size_t>(blockDim.x * blockIdx.x + threadIdx.x);
  if (data_index < num_data) {
    if (!USE_WEIGHT) {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>((diff > 0.0f) - (diff < 0.0f));
      cuda_out_hessians[data_index] = 1.0f;
    } else {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      const score_t weight = static_cast<score_t>(cuda_weights[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>((diff > 0.0f) - (diff < 0.0f)) * weight;
      cuda_out_hessians[data_index] = weight;
    }
  }
}

void CUDARegressionL1loss::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  if (cuda_weights_ == nullptr) {
    GetGradientsKernel_RegressionL1<false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, nullptr, num_data_, gradients, hessians);
  } else {
    GetGradientsKernel_RegressionL1<true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, cuda_weights_, num_data_, gradients, hessians);
  }
}

template <bool USE_WEIGHT>
__global__ void RenewTreeOutputCUDAKernel_RegressionL1(
  const double* score,
  const label_t* label,
  const label_t* weight,
  double* residual_buffer,
  label_t* weight_by_leaf,
  double* weight_prefix_sum_buffer,
  const data_size_t* data_indices_in_leaf,
  const data_size_t* num_data_in_leaf,
  const data_size_t* data_start_in_leaf,
  data_size_t* data_indices_buffer,
  double* leaf_value) {
  const int leaf_index = static_cast<int>(blockIdx.x);
  const data_size_t data_start = data_start_in_leaf[leaf_index];
  const data_size_t num_data = num_data_in_leaf[leaf_index];
  data_size_t* data_indices_buffer_pointer = data_indices_buffer + data_start;
  const label_t* weight_by_leaf_pointer = weight_by_leaf + data_start;
  double* weight_prefix_sum_buffer_pointer = weight_prefix_sum_buffer + data_start;
  const double* residual_buffer_pointer = residual_buffer + data_start;
  const double alpha = 0.5f;
  for (data_size_t inner_data_index = data_start + static_cast<data_size_t>(threadIdx.x); inner_data_index < data_start + num_data; inner_data_index += static_cast<data_size_t>(blockDim.x)) {
    const data_size_t data_index = data_indices_in_leaf[inner_data_index];
    const label_t data_label = label[data_index];
    const double data_score = score[data_index];
    residual_buffer[inner_data_index] = static_cast<double>(data_label) - data_score;
    if (USE_WEIGHT) {
      weight_by_leaf[inner_data_index] = weight[data_index];
    }
  }
  __syncthreads();
  const double renew_leaf_value = PercentileDevice<double, data_size_t, label_t, double, false, USE_WEIGHT>(
    residual_buffer_pointer, weight_by_leaf_pointer, data_indices_buffer_pointer,
    weight_prefix_sum_buffer_pointer, alpha, num_data);
  if (threadIdx.x == 0) {
    leaf_value[leaf_index] = renew_leaf_value;
  }
}

void CUDARegressionL1loss::LaunchRenewTreeOutputCUDAKernel(
  const double* score,
  const data_size_t* data_indices_in_leaf,
  const data_size_t* num_data_in_leaf,
  const data_size_t* data_start_in_leaf,
  const int num_leaves,
  double* leaf_value) const {
  if (cuda_weights_ == nullptr) {
    RenewTreeOutputCUDAKernel_RegressionL1<false><<<num_leaves, GET_GRADIENTS_BLOCK_SIZE_REGRESSION / 2>>>(
      score,
      cuda_labels_,
      cuda_weights_,
      cuda_residual_buffer_.RawData(),
      cuda_weight_by_leaf_buffer_.RawData(),
      cuda_weights_prefix_sum_.RawData(),
      data_indices_in_leaf,
      num_data_in_leaf,
      data_start_in_leaf,
      cuda_data_indices_buffer_.RawData(),
      leaf_value);
  } else {
    RenewTreeOutputCUDAKernel_RegressionL1<true><<<num_leaves, GET_GRADIENTS_BLOCK_SIZE_REGRESSION / 4>>>(
      score,
      cuda_labels_,
      cuda_weights_,
      cuda_residual_buffer_.RawData(),
      cuda_weight_by_leaf_buffer_.RawData(),
      cuda_weights_prefix_sum_.RawData(),
      data_indices_in_leaf,
      num_data_in_leaf,
      data_start_in_leaf,
      cuda_data_indices_buffer_.RawData(),
      leaf_value);
  }
  SynchronizeCUDADevice(__FILE__, __LINE__);
}


template <bool USE_WEIGHT>
__global__ void GetGradientsKernel_Huber(const double* cuda_scores, const label_t* cuda_labels, const label_t* cuda_weights, const data_size_t num_data,
  const double alpha, score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  const data_size_t data_index = static_cast<data_size_t>(blockDim.x * blockIdx.x + threadIdx.x);
  if (data_index < num_data) {
    if (!USE_WEIGHT) {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      if (fabs(diff) <= alpha) {
        cuda_out_gradients[data_index] = static_cast<score_t>(diff);
      } else {
        const score_t sign = static_cast<score_t>((diff > 0.0f) - (diff < 0.0f));
        cuda_out_gradients[data_index] = static_cast<score_t>(sign * alpha);
      }
      cuda_out_hessians[data_index] = 1.0f;
    } else {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      const score_t weight = static_cast<score_t>(cuda_weights[data_index]);
      if (fabs(diff) <= alpha) {
        cuda_out_gradients[data_index] = static_cast<score_t>(diff) * weight;
      } else {
        const score_t sign = static_cast<score_t>((diff > 0.0f) - (diff < 0.0f));
        cuda_out_gradients[data_index] = static_cast<score_t>(sign * alpha) * weight;
      }
      cuda_out_hessians[data_index] = weight;
    }
  }
}

void CUDARegressionHuberLoss::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  if (cuda_weights_ == nullptr) {
    GetGradientsKernel_Huber<false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, nullptr, num_data_, alpha_, gradients, hessians);
  } else {
    GetGradientsKernel_Huber<true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, cuda_weights_, num_data_, alpha_, gradients, hessians);
  }
}


template <bool USE_WEIGHT>
__global__ void GetGradientsKernel_Fair(const double* cuda_scores, const label_t* cuda_labels, const label_t* cuda_weights, const data_size_t num_data,
  const double c, score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  const data_size_t data_index = static_cast<data_size_t>(blockDim.x * blockIdx.x + threadIdx.x);
  if (data_index < num_data) {
    if (!USE_WEIGHT) {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>(c * diff / (fabs(diff) + c));
      cuda_out_hessians[data_index] = static_cast<score_t>(c * c / ((fabs(diff) + c) * (fabs(diff) + c)));
    } else {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      const score_t weight = static_cast<score_t>(cuda_weights[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>(c * diff / (fabs(diff) + c) * weight);
      cuda_out_hessians[data_index] = static_cast<score_t>(c * c / ((fabs(diff) + c) * (fabs(diff) + c)) * weight);
    }
  }
}

void CUDARegressionFairLoss::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  if (cuda_weights_ == nullptr) {
    GetGradientsKernel_Fair<false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, nullptr, num_data_, c_, gradients, hessians);
  } else {
    GetGradientsKernel_Fair<true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, cuda_weights_, num_data_, c_, gradients, hessians);
  }
}


}  // namespace LightGBM

#endif  // USE_CUDA_EXP
