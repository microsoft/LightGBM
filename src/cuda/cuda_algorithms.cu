
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for license information.
 */

#ifdef USE_CUDA_EXP

#include <LightGBM/cuda/cuda_algorithms.hpp>

namespace LightGBM {

template <typename T>
__global__ void ShufflePrefixSumGlobalKernel(T* values, size_t len, T* block_prefix_sum_buffer) {
  __shared__ T shared_mem_buffer[32];
  const size_t index = static_cast<size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  T value = 0;
  if (index < len) {
    value = values[index];
  }
  const T prefix_sum_value = ShufflePrefixSum<T>(value, shared_mem_buffer);
  values[index] = prefix_sum_value;
  if (threadIdx.x == blockDim.x - 1) {
    block_prefix_sum_buffer[blockIdx.x] = prefix_sum_value;
  }
}

template <typename T>
__global__ void ShufflePrefixSumGlobalReduceBlockKernel(T* block_prefix_sum_buffer, int num_blocks) {
  __shared__ T shared_mem_buffer[32];
  const int num_blocks_per_thread = (num_blocks + GLOBAL_PREFIX_SUM_BLOCK_SIZE - 2) / (GLOBAL_PREFIX_SUM_BLOCK_SIZE - 1);
  int thread_block_start = threadIdx.x == 0 ? 0 : (threadIdx.x - 1) * num_blocks_per_thread;
  int thread_block_end = threadIdx.x == 0 ? 0 : min(thread_block_start + num_blocks_per_thread, num_blocks);
  T base = 0;
  for (int block_index = thread_block_start; block_index < thread_block_end; ++block_index) {
    base += block_prefix_sum_buffer[block_index];
  }
  base = ShufflePrefixSum<T>(base, shared_mem_buffer);
  thread_block_start = threadIdx.x == blockDim.x - 1 ? 0 : threadIdx.x * num_blocks_per_thread;
  thread_block_end = threadIdx.x == blockDim.x - 1 ? 0 : min(thread_block_start + num_blocks_per_thread, num_blocks);
  for (int block_index = thread_block_start + 1; block_index < thread_block_end; ++block_index) {
    block_prefix_sum_buffer[block_index] += block_prefix_sum_buffer[block_index - 1];
  }
  for (int block_index = thread_block_start; block_index < thread_block_end; ++block_index) {
    block_prefix_sum_buffer[block_index] += base;
  }
}

template <typename T>
__global__ void ShufflePrefixSumGlobalAddBase(size_t len, const T* block_prefix_sum_buffer, T* values) {
  const T base = blockIdx.x == 0 ? 0 : block_prefix_sum_buffer[blockIdx.x - 1];
  const size_t index = static_cast<size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (index < len) {
    values[index] += base;
  }
}

template <typename T>
void ShufflePrefixSumGlobalInner(T* values, size_t len, T* block_prefix_sum_buffer) {
  const int num_blocks = (static_cast<int>(len) + GLOBAL_PREFIX_SUM_BLOCK_SIZE - 1) / GLOBAL_PREFIX_SUM_BLOCK_SIZE;
  ShufflePrefixSumGlobalKernel<<<num_blocks, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(values, len, block_prefix_sum_buffer);
  ShufflePrefixSumGlobalReduceBlockKernel<<<1, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(block_prefix_sum_buffer, num_blocks);
  ShufflePrefixSumGlobalAddBase<<<num_blocks, GLOBAL_PREFIX_SUM_BLOCK_SIZE>>>(len, block_prefix_sum_buffer, values);
}

template <>
void ShufflePrefixSumGlobal(uint16_t* values, size_t len, uint16_t* block_prefix_sum_buffer) {
  ShufflePrefixSumGlobalInner<uint16_t>(values, len, block_prefix_sum_buffer);
}

template <>
void ShufflePrefixSumGlobal(uint32_t* values, size_t len, uint32_t* block_prefix_sum_buffer) {
  ShufflePrefixSumGlobalInner<uint32_t>(values, len, block_prefix_sum_buffer);
}

template <>
void ShufflePrefixSumGlobal(uint64_t* values, size_t len, uint64_t* block_prefix_sum_buffer) {
  ShufflePrefixSumGlobalInner<uint64_t>(values, len, block_prefix_sum_buffer);
}

}  // namespace LightGBM

#endif  // USE_CUDA_EXP
