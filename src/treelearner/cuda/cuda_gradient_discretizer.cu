
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include <algorithm>

#include <LightGBM/cuda/cuda_algorithms.hpp>

#include "cuda_gradient_discretizer.hpp"

namespace LightGBM {

__global__ void ReduceMinMaxKernel(
  const data_size_t num_data,
  const score_t* input_gradients,
  const score_t* input_hessians,
  score_t* grad_min_block_buffer,
  score_t* grad_max_block_buffer,
  score_t* hess_min_block_buffer,
  score_t* hess_max_block_buffer) {
  __shared__ score_t shared_mem_buffer[32];
  const data_size_t index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  score_t grad_max_val = kMinScore;
  score_t grad_min_val = kMaxScore;
  score_t hess_max_val = kMinScore;
  score_t hess_min_val = kMaxScore;
  if (index < num_data) {
    grad_max_val = input_gradients[index];
    grad_min_val = input_gradients[index];
    hess_max_val = input_hessians[index];
    hess_min_val = input_hessians[index];
  }
  grad_min_val = ShuffleReduceMin<score_t>(grad_min_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  grad_max_val = ShuffleReduceMax<score_t>(grad_max_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  hess_min_val = ShuffleReduceMin<score_t>(hess_min_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  hess_max_val = ShuffleReduceMax<score_t>(hess_max_val, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    grad_min_block_buffer[blockIdx.x] = grad_min_val;
    grad_max_block_buffer[blockIdx.x] = grad_max_val;
    hess_min_block_buffer[blockIdx.x] = hess_min_val;
    hess_max_block_buffer[blockIdx.x] = hess_max_val;
  }
}

__global__ void ReduceBlockMinMaxKernel(
  const int num_blocks,
  const int grad_discretize_bins,
  score_t* grad_min_block_buffer,
  score_t* grad_max_block_buffer,
  score_t* hess_min_block_buffer,
  score_t* hess_max_block_buffer) {
  __shared__ score_t shared_mem_buffer[32];
  score_t grad_max_val = kMinScore;
  score_t grad_min_val = kMaxScore;
  score_t hess_max_val = kMinScore;
  score_t hess_min_val = kMaxScore;
  for (int block_index = static_cast<int>(threadIdx.x); block_index < num_blocks; block_index += static_cast<int>(blockDim.x)) {
    grad_min_val = min(grad_min_val, grad_min_block_buffer[block_index]);
    grad_max_val = max(grad_max_val, grad_max_block_buffer[block_index]);
    hess_min_val = min(hess_min_val, hess_min_block_buffer[block_index]);
    hess_max_val = max(hess_max_val, hess_max_block_buffer[block_index]);
  }
  grad_min_val = ShuffleReduceMin<score_t>(grad_min_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  grad_max_val = ShuffleReduceMax<score_t>(grad_max_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  hess_max_val = ShuffleReduceMax<score_t>(hess_max_val, shared_mem_buffer, blockDim.x);
  __syncthreads();
  hess_max_val = ShuffleReduceMax<score_t>(hess_max_val, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    const score_t grad_abs_max = max(fabs(grad_min_val), fabs(grad_max_val));
    const score_t hess_abs_max = max(fabs(hess_min_val), fabs(hess_max_val));
    grad_min_block_buffer[0] = 1.0f / (grad_abs_max / (grad_discretize_bins / 2));
    grad_max_block_buffer[0] = (grad_abs_max / (grad_discretize_bins / 2));
    hess_min_block_buffer[0] = 1.0f / (hess_abs_max / (grad_discretize_bins));
    hess_max_block_buffer[0] = (hess_abs_max / (grad_discretize_bins));
  }
}

template <bool STOCHASTIC_ROUNDING>
__global__ void DiscretizeGradientsKernel(
  const data_size_t num_data,
  const score_t* input_gradients,
  const score_t* input_hessians,
  const score_t* grad_scale_ptr,
  const score_t* hess_scale_ptr,
  const int iter,
  const int* random_values_use_start,
  const score_t* gradient_random_values,
  const score_t* hessian_random_values,
  const int grad_discretize_bins,
  int8_t* output_gradients_and_hessians) {
  const int start = random_values_use_start[iter];
  const data_size_t index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  const score_t grad_scale = *grad_scale_ptr;
  const score_t hess_scale = *hess_scale_ptr;
  int16_t* output_gradients_and_hessians_ptr = reinterpret_cast<int16_t*>(output_gradients_and_hessians);
  if (index < num_data) {
    if (STOCHASTIC_ROUNDING) {
      const data_size_t index_offset = (index + start) % num_data;
      const score_t gradient = input_gradients[index];
      const score_t hessian = input_hessians[index];
      const score_t gradient_random_value = gradient_random_values[index_offset];
      const score_t hessian_random_value = hessian_random_values[index_offset];
      output_gradients_and_hessians_ptr[2 * index + 1] = gradient > 0.0f ?
        static_cast<int16_t>(gradient * grad_scale + gradient_random_value) :
        static_cast<int16_t>(gradient * grad_scale - gradient_random_value);
      output_gradients_and_hessians_ptr[2 * index] = static_cast<int16_t>(hessian * hess_scale + hessian_random_value);
    } else {
      const score_t gradient = input_gradients[index];
      const score_t hessian = input_hessians[index];
      output_gradients_and_hessians_ptr[2 * index + 1] = gradient > 0.0f ?
        static_cast<int16_t>(gradient * grad_scale + 0.5) :
        static_cast<int16_t>(gradient * grad_scale - 0.5);
      output_gradients_and_hessians_ptr[2 * index] = static_cast<int16_t>(hessian * hess_scale + 0.5);
    }
  }
}

void CUDAGradientDiscretizer::DiscretizeGradients(
  const data_size_t num_data,
  const score_t* input_gradients,
  const score_t* input_hessians) {
  ReduceMinMaxKernel<<<num_reduce_blocks_, CUDA_GRADIENT_DISCRETIZER_BLOCK_SIZE>>>(
    num_data, input_gradients, input_hessians,
    grad_min_block_buffer_.RawData(),
    grad_max_block_buffer_.RawData(),
    hess_min_block_buffer_.RawData(),
    hess_max_block_buffer_.RawData());
    SynchronizeCUDADevice(__FILE__, __LINE__);
  ReduceBlockMinMaxKernel<<<1, CUDA_GRADIENT_DISCRETIZER_BLOCK_SIZE>>>(
    num_reduce_blocks_,
    num_grad_quant_bins_,
    grad_min_block_buffer_.RawData(),
    grad_max_block_buffer_.RawData(),
    hess_min_block_buffer_.RawData(),
    hess_max_block_buffer_.RawData());
    SynchronizeCUDADevice(__FILE__, __LINE__);

  #define DiscretizeGradientsKernel_ARGS \
    num_data, \
    input_gradients, \
    input_hessians, \
    grad_min_block_buffer_.RawData(), \
    hess_min_block_buffer_.RawData(), \
    iter_, \
    random_values_use_start_.RawData(), \
    gradient_random_values_.RawData(), \
    hessian_random_values_.RawData(), \
    num_grad_quant_bins_, \
    discretized_gradients_and_hessians_.RawData()

  if (stochastic_rounding_) {
    DiscretizeGradientsKernel<true><<<num_reduce_blocks_, CUDA_GRADIENT_DISCRETIZER_BLOCK_SIZE>>>(DiscretizeGradientsKernel_ARGS);
  } else {
    DiscretizeGradientsKernel<false><<<num_reduce_blocks_, CUDA_GRADIENT_DISCRETIZER_BLOCK_SIZE>>>(DiscretizeGradientsKernel_ARGS);
  }
  SynchronizeCUDADevice(__FILE__, __LINE__);
  ++iter_;
}

}  // namespace LightGBM

#endif  // USE_CUDA
