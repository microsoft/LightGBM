
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for license information.
 */


#ifdef USE_CUDA

#include <LightGBM/cuda/cuda_column_data.hpp>

#define COPY_SUBROW_BLOCK_SIZE_COLUMN_DATA (1024)

namespace LightGBM {

__global__ void CopySubrowKernel_ColumnData(
  void* const* in_cuda_data_by_column,
  const uint8_t* cuda_column_bit_type,
  const data_size_t* cuda_used_indices,
  const data_size_t num_used_indices,
  const int num_column,
  void** out_cuda_data_by_column) {
  const data_size_t local_data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (local_data_index < num_used_indices) {
    for (int column_index = 0; column_index < num_column; ++column_index) {
      const void* in_column_data = in_cuda_data_by_column[column_index];
      void* out_column_data = out_cuda_data_by_column[column_index];
      const uint8_t bit_type = cuda_column_bit_type[column_index];
      if (bit_type == 8) {
        const uint8_t* true_in_column_data = reinterpret_cast<const uint8_t*>(in_column_data);
        uint8_t* true_out_column_data = reinterpret_cast<uint8_t*>(out_column_data);
        const data_size_t global_data_index = cuda_used_indices[local_data_index];
        true_out_column_data[local_data_index] = true_in_column_data[global_data_index];
      } else if (bit_type == 16) {
        const uint16_t* true_in_column_data = reinterpret_cast<const uint16_t*>(in_column_data);
        uint16_t* true_out_column_data = reinterpret_cast<uint16_t*>(out_column_data);
        const data_size_t global_data_index = cuda_used_indices[local_data_index];
        true_out_column_data[local_data_index] = true_in_column_data[global_data_index];
      } else if (bit_type == 32) {
        const uint32_t* true_in_column_data = reinterpret_cast<const uint32_t*>(in_column_data);
        uint32_t* true_out_column_data = reinterpret_cast<uint32_t*>(out_column_data);
        const data_size_t global_data_index = cuda_used_indices[local_data_index];
        true_out_column_data[local_data_index] = true_in_column_data[global_data_index];
      }
    }
  }
}

void CUDAColumnData::LaunchCopySubrowKernel(void* const* in_cuda_data_by_column) {
  const int num_blocks = (num_used_indices_ + COPY_SUBROW_BLOCK_SIZE_COLUMN_DATA - 1) / COPY_SUBROW_BLOCK_SIZE_COLUMN_DATA;
  CopySubrowKernel_ColumnData<<<num_blocks, COPY_SUBROW_BLOCK_SIZE_COLUMN_DATA>>>(
    in_cuda_data_by_column,
    cuda_column_bit_type_,
    cuda_used_indices_,
    num_used_indices_,
    num_columns_,
    cuda_data_by_column_);
}

}  // namespace LightGBM

#endif  // USE_CUDA
