
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include "cuda_regression_objective.hpp"
#include <LightGBM/cuda/cuda_algorithms.hpp>

namespace LightGBM {

template <typename HOST_OBJECTIVE>
void CUDARegressionObjectiveInterface<HOST_OBJECTIVE>::Init(const Metadata& metadata, data_size_t num_data) {
  CUDAObjectiveInterface<HOST_OBJECTIVE>::Init(metadata, num_data);
  const data_size_t num_get_gradients_blocks = (this->num_data_ + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  cuda_block_buffer_.Resize(static_cast<size_t>(num_get_gradients_blocks));
  if (this->sqrt_) {
    cuda_trans_label_.Resize(this->trans_label_.size());
    CopyFromHostToCUDADevice<label_t>(cuda_trans_label_.RawData(), this->trans_label_.data(), this->trans_label_.size(), __FILE__, __LINE__);
    this->cuda_labels_ = cuda_trans_label_.RawData();
  }
}

template void CUDARegressionObjectiveInterface<RegressionL2loss>::Init(const Metadata& metadata, data_size_t num_data);
template void CUDARegressionObjectiveInterface<RegressionL1loss>::Init(const Metadata& metadata, data_size_t num_data);
template void CUDARegressionObjectiveInterface<RegressionHuberLoss>::Init(const Metadata& metadata, data_size_t num_data);
template void CUDARegressionObjectiveInterface<RegressionFairLoss>::Init(const Metadata& metadata, data_size_t num_data);
template void CUDARegressionObjectiveInterface<RegressionPoissonLoss>::Init(const Metadata& metadata, data_size_t num_data);
template void CUDARegressionObjectiveInterface<RegressionQuantileloss>::Init(const Metadata& metadata, data_size_t num_data);

template <typename HOST_OBJECTIVE>
double CUDARegressionObjectiveInterface<HOST_OBJECTIVE>::LaunchCalcInitScoreKernel(const int /*class_id*/) const {
  double label_sum = 0.0f, weight_sum = 0.0f;
  if (this->cuda_weights_ == nullptr) {
    ShuffleReduceSumGlobal<label_t, double>(this->cuda_labels_,
      static_cast<size_t>(this->num_data_), cuda_block_buffer_.RawData());
    CopyFromCUDADeviceToHost<double>(&label_sum, cuda_block_buffer_.RawData(), 1, __FILE__, __LINE__);
    weight_sum = static_cast<double>(this->num_data_);
  } else {
    ShuffleReduceDotProdGlobal<label_t, double>(this->cuda_labels_,
      this->cuda_weights_, static_cast<size_t>(this->num_data_), cuda_block_buffer_.RawData());
    CopyFromCUDADeviceToHost<double>(&label_sum, cuda_block_buffer_.RawData(), 1, __FILE__, __LINE__);
    ShuffleReduceSumGlobal<label_t, double>(this->cuda_weights_,
      static_cast<size_t>(this->num_data_), cuda_block_buffer_.RawData());
    CopyFromCUDADeviceToHost<double>(&weight_sum, cuda_block_buffer_.RawData(), 1, __FILE__, __LINE__);
  }
  return label_sum / weight_sum;
}

template double CUDARegressionObjectiveInterface<RegressionL2loss>::LaunchCalcInitScoreKernel(const int class_id) const;
template double CUDARegressionObjectiveInterface<RegressionL1loss>::LaunchCalcInitScoreKernel(const int class_id) const;
template double CUDARegressionObjectiveInterface<RegressionHuberLoss>::LaunchCalcInitScoreKernel(const int class_id) const;
template double CUDARegressionObjectiveInterface<RegressionFairLoss>::LaunchCalcInitScoreKernel(const int class_id) const;
template double CUDARegressionObjectiveInterface<RegressionPoissonLoss>::LaunchCalcInitScoreKernel(const int class_id) const;
template double CUDARegressionObjectiveInterface<RegressionQuantileloss>::LaunchCalcInitScoreKernel(const int class_id) const;

__global__ void ConvertOutputCUDAKernel_Regression(const bool sqrt, const data_size_t num_data, const double* input, double* output) {
  const int data_index = static_cast<data_size_t>(blockIdx.x * blockDim.x + threadIdx.x);
  if (data_index < num_data) {
    if (sqrt) {
      const double sign = input[data_index] >= 0.0f ? 1 : -1;
      output[data_index] = sign * input[data_index] * input[data_index];
    } else {
      output[data_index] = input[data_index];
    }
  }
}

const double* CUDARegressionL2loss::LaunchConvertOutputCUDAKernel(const data_size_t num_data, const double* input, double* output) const {
  const int num_blocks = (num_data + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  if (sqrt_) {
    ConvertOutputCUDAKernel_Regression<<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(sqrt_, num_data, input, output);
    return output;
  } else {
    return input;
  }
}

template <bool USE_WEIGHT>
__global__ void GetGradientsKernel_RegressionL2(const double* cuda_scores, const label_t* cuda_labels, const label_t* cuda_weights, const data_size_t num_data,
  score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  const data_size_t data_index = static_cast<data_size_t>(blockDim.x * blockIdx.x + threadIdx.x);
  if (data_index < num_data) {
    if (!USE_WEIGHT) {
      cuda_out_gradients[data_index] = static_cast<score_t>(cuda_scores[data_index] - cuda_labels[data_index]);
      cuda_out_hessians[data_index] = 1.0f;
    } else {
      const score_t weight = static_cast<score_t>(cuda_weights[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>(cuda_scores[data_index] - cuda_labels[data_index]) * weight;
      cuda_out_hessians[data_index] = weight;
    }
  }
}

void CUDARegressionL2loss::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  if (cuda_weights_ == nullptr) {
    GetGradientsKernel_RegressionL2<false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, nullptr, num_data_, gradients, hessians);
  } else {
    GetGradientsKernel_RegressionL2<true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, cuda_weights_, num_data_, gradients, hessians);
  }
}


double CUDARegressionL1loss::LaunchCalcInitScoreKernel(const int /*class_id*/) const {
  const double alpha = 0.5f;
  if (cuda_weights_ == nullptr) {
    PercentileGlobal<label_t, data_size_t, label_t, double, false, false>(
      cuda_labels_, nullptr, cuda_data_indices_buffer_.RawData(), nullptr, nullptr, alpha, num_data_, cuda_percentile_result_.RawData());
  } else {
    PercentileGlobal<label_t, data_size_t, label_t, double, false, true>(
      cuda_labels_, cuda_weights_, cuda_data_indices_buffer_.RawData(), cuda_weights_prefix_sum_.RawData(),
      cuda_weights_prefix_sum_buffer_.RawData(), alpha, num_data_, cuda_percentile_result_.RawData());
  }
  label_t percentile_result = 0.0f;
  CopyFromCUDADeviceToHost<label_t>(&percentile_result, cuda_percentile_result_.RawData(), 1, __FILE__, __LINE__);
  SynchronizeCUDADevice(__FILE__, __LINE__);
  return static_cast<label_t>(percentile_result);
}

template <bool USE_WEIGHT>
__global__ void GetGradientsKernel_RegressionL1(const double* cuda_scores, const label_t* cuda_labels, const label_t* cuda_weights, const data_size_t num_data,
  score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  const data_size_t data_index = static_cast<data_size_t>(blockDim.x * blockIdx.x + threadIdx.x);
  if (data_index < num_data) {
    if (!USE_WEIGHT) {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>((diff > 0.0f) - (diff < 0.0f));
      cuda_out_hessians[data_index] = 1.0f;
    } else {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      const score_t weight = static_cast<score_t>(cuda_weights[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>((diff > 0.0f) - (diff < 0.0f)) * weight;
      cuda_out_hessians[data_index] = weight;
    }
  }
}

void CUDARegressionL1loss::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  if (cuda_weights_ == nullptr) {
    GetGradientsKernel_RegressionL1<false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, nullptr, num_data_, gradients, hessians);
  } else {
    GetGradientsKernel_RegressionL1<true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, cuda_weights_, num_data_, gradients, hessians);
  }
}

template <bool USE_WEIGHT>
__global__ void RenewTreeOutputCUDAKernel_RegressionL1(
  const double* score,
  const label_t* label,
  const label_t* weight,
  double* residual_buffer,
  label_t* weight_by_leaf,
  double* weight_prefix_sum_buffer,
  const data_size_t* data_indices_in_leaf,
  const data_size_t* num_data_in_leaf,
  const data_size_t* data_start_in_leaf,
  data_size_t* data_indices_buffer,
  double* leaf_value) {
  const int leaf_index = static_cast<int>(blockIdx.x);
  const data_size_t data_start = data_start_in_leaf[leaf_index];
  const data_size_t num_data = num_data_in_leaf[leaf_index];
  data_size_t* data_indices_buffer_pointer = data_indices_buffer + data_start;
  const label_t* weight_by_leaf_pointer = weight_by_leaf + data_start;
  double* weight_prefix_sum_buffer_pointer = weight_prefix_sum_buffer + data_start;
  const double* residual_buffer_pointer = residual_buffer + data_start;
  const double alpha = 0.5f;
  for (data_size_t inner_data_index = data_start + static_cast<data_size_t>(threadIdx.x);
    inner_data_index < data_start + num_data; inner_data_index += static_cast<data_size_t>(blockDim.x)) {
    const data_size_t data_index = data_indices_in_leaf[inner_data_index];
    const label_t data_label = label[data_index];
    const double data_score = score[data_index];
    residual_buffer[inner_data_index] = static_cast<double>(data_label) - data_score;
    if (USE_WEIGHT) {
      weight_by_leaf[inner_data_index] = weight[data_index];
    }
  }
  __syncthreads();
  const double renew_leaf_value = PercentileDevice<double, data_size_t, label_t, double, false, USE_WEIGHT>(
    residual_buffer_pointer, weight_by_leaf_pointer, data_indices_buffer_pointer,
    weight_prefix_sum_buffer_pointer, alpha, num_data);
  if (threadIdx.x == 0) {
    leaf_value[leaf_index] = renew_leaf_value;
  }
}

void CUDARegressionL1loss::LaunchRenewTreeOutputCUDAKernel(
  const double* score,
  const data_size_t* data_indices_in_leaf,
  const data_size_t* num_data_in_leaf,
  const data_size_t* data_start_in_leaf,
  const int num_leaves,
  double* leaf_value) const {
  if (cuda_weights_ == nullptr) {
    RenewTreeOutputCUDAKernel_RegressionL1<false><<<num_leaves, GET_GRADIENTS_BLOCK_SIZE_REGRESSION / 2>>>(
      score,
      cuda_labels_,
      cuda_weights_,
      cuda_residual_buffer_.RawData(),
      cuda_weight_by_leaf_buffer_.RawData(),
      cuda_weights_prefix_sum_.RawData(),
      data_indices_in_leaf,
      num_data_in_leaf,
      data_start_in_leaf,
      cuda_data_indices_buffer_.RawData(),
      leaf_value);
  } else {
    RenewTreeOutputCUDAKernel_RegressionL1<true><<<num_leaves, GET_GRADIENTS_BLOCK_SIZE_REGRESSION / 4>>>(
      score,
      cuda_labels_,
      cuda_weights_,
      cuda_residual_buffer_.RawData(),
      cuda_weight_by_leaf_buffer_.RawData(),
      cuda_weights_prefix_sum_.RawData(),
      data_indices_in_leaf,
      num_data_in_leaf,
      data_start_in_leaf,
      cuda_data_indices_buffer_.RawData(),
      leaf_value);
  }
  SynchronizeCUDADevice(__FILE__, __LINE__);
}


template <bool USE_WEIGHT>
__global__ void GetGradientsKernel_Huber(const double* cuda_scores, const label_t* cuda_labels, const label_t* cuda_weights, const data_size_t num_data,
  const double alpha, score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  const data_size_t data_index = static_cast<data_size_t>(blockDim.x * blockIdx.x + threadIdx.x);
  if (data_index < num_data) {
    if (!USE_WEIGHT) {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      if (fabs(diff) <= alpha) {
        cuda_out_gradients[data_index] = static_cast<score_t>(diff);
      } else {
        const score_t sign = static_cast<score_t>((diff > 0.0f) - (diff < 0.0f));
        cuda_out_gradients[data_index] = static_cast<score_t>(sign * alpha);
      }
      cuda_out_hessians[data_index] = 1.0f;
    } else {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      const score_t weight = static_cast<score_t>(cuda_weights[data_index]);
      if (fabs(diff) <= alpha) {
        cuda_out_gradients[data_index] = static_cast<score_t>(diff) * weight;
      } else {
        const score_t sign = static_cast<score_t>((diff > 0.0f) - (diff < 0.0f));
        cuda_out_gradients[data_index] = static_cast<score_t>(sign * alpha) * weight;
      }
      cuda_out_hessians[data_index] = weight;
    }
  }
}

void CUDARegressionHuberLoss::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  if (cuda_weights_ == nullptr) {
    GetGradientsKernel_Huber<false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, nullptr, num_data_, alpha_, gradients, hessians);
  } else {
    GetGradientsKernel_Huber<true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, cuda_weights_, num_data_, alpha_, gradients, hessians);
  }
}


template <bool USE_WEIGHT>
__global__ void GetGradientsKernel_Fair(const double* cuda_scores, const label_t* cuda_labels, const label_t* cuda_weights, const data_size_t num_data,
  const double c, score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  const data_size_t data_index = static_cast<data_size_t>(blockDim.x * blockIdx.x + threadIdx.x);
  if (data_index < num_data) {
    if (!USE_WEIGHT) {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>(c * diff / (fabs(diff) + c));
      cuda_out_hessians[data_index] = static_cast<score_t>(c * c / ((fabs(diff) + c) * (fabs(diff) + c)));
    } else {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      const score_t weight = static_cast<score_t>(cuda_weights[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>(c * diff / (fabs(diff) + c) * weight);
      cuda_out_hessians[data_index] = static_cast<score_t>(c * c / ((fabs(diff) + c) * (fabs(diff) + c)) * weight);
    }
  }
}

void CUDARegressionFairLoss::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  if (cuda_weights_ == nullptr) {
    GetGradientsKernel_Fair<false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, nullptr, num_data_, c_, gradients, hessians);
  } else {
    GetGradientsKernel_Fair<true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, cuda_weights_, num_data_, c_, gradients, hessians);
  }
}

void CUDARegressionPoissonLoss::LaunchCheckLabelKernel() const {
  ShuffleReduceSumGlobal<label_t, double>(cuda_labels_, static_cast<size_t>(num_data_), cuda_block_buffer_.RawData());
  double label_sum = 0.0f;
  CopyFromCUDADeviceToHost<double>(&label_sum, cuda_block_buffer_.RawData(), 1, __FILE__, __LINE__);

  ShuffleReduceMinGlobal<label_t, double>(cuda_labels_, static_cast<size_t>(num_data_), cuda_block_buffer_.RawData());
  double label_min = 0.0f;
  CopyFromCUDADeviceToHost<double>(&label_min, cuda_block_buffer_.RawData(), 1, __FILE__, __LINE__);

  if (label_min < 0.0f) {
    Log::Fatal("[%s]: at least one target label is negative", GetName());
  }
  if (label_sum == 0.0f) {
    Log::Fatal("[%s]: sum of labels is zero", GetName());
  }
}

template <bool USE_WEIGHT>
__global__ void GetGradientsKernel_Poisson(const double* cuda_scores, const label_t* cuda_labels, const label_t* cuda_weights, const data_size_t num_data,
  const double max_delta_step, score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  const data_size_t data_index = static_cast<data_size_t>(blockDim.x * blockIdx.x + threadIdx.x);
  const double exp_max_delta_step = std::exp(max_delta_step);
  if (data_index < num_data) {
    if (!USE_WEIGHT) {
      const double exp_score = exp(cuda_scores[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>(exp_score - cuda_labels[data_index]);
      cuda_out_hessians[data_index] = static_cast<score_t>(exp_score * exp_max_delta_step);
    } else {
      const double exp_score = exp(cuda_scores[data_index]);
      const score_t weight = static_cast<score_t>(cuda_weights[data_index]);
      cuda_out_gradients[data_index] = static_cast<score_t>((exp_score - cuda_labels[data_index]) * weight);
      cuda_out_hessians[data_index] = static_cast<score_t>(exp_score * exp_max_delta_step * weight);
    }
  }
}

void CUDARegressionPoissonLoss::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  if (cuda_weights_ == nullptr) {
    GetGradientsKernel_Poisson<false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(
      score, cuda_labels_, nullptr, num_data_, max_delta_step_, gradients, hessians);
  } else {
    GetGradientsKernel_Poisson<true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(
      score, cuda_labels_, cuda_weights_, num_data_, max_delta_step_, gradients, hessians);
  }
}

__global__ void ConvertOutputCUDAKernel_Regression_Poisson(const data_size_t num_data, const double* input, double* output) {
  const int data_index = static_cast<data_size_t>(blockIdx.x * blockDim.x + threadIdx.x);
  if (data_index < num_data) {
    output[data_index] = exp(input[data_index]);
  }
}

const double* CUDARegressionPoissonLoss::LaunchConvertOutputCUDAKernel(const data_size_t num_data, const double* input, double* output) const {
  const int num_blocks = (num_data + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  ConvertOutputCUDAKernel_Regression_Poisson<<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(num_data, input, output);
  return output;
}


double CUDARegressionQuantileloss::LaunchCalcInitScoreKernel(const int /*class_id*/) const {
  if (cuda_weights_ == nullptr) {
    PercentileGlobal<label_t, data_size_t, label_t, double, false, false>(
      cuda_labels_, nullptr, cuda_data_indices_buffer_.RawData(), nullptr, nullptr, alpha_, num_data_, cuda_percentile_result_.RawData());
  } else {
    PercentileGlobal<label_t, data_size_t, label_t, double, false, true>(
      cuda_labels_, cuda_weights_, cuda_data_indices_buffer_.RawData(), cuda_weights_prefix_sum_.RawData(),
      cuda_weights_prefix_sum_buffer_.RawData(), alpha_, num_data_, cuda_percentile_result_.RawData());
  }
  label_t percentile_result = 0.0f;
  CopyFromCUDADeviceToHost<label_t>(&percentile_result, cuda_percentile_result_.RawData(), 1, __FILE__, __LINE__);
  SynchronizeCUDADevice(__FILE__, __LINE__);
  return static_cast<label_t>(percentile_result);
}

template <bool USE_WEIGHT>
__global__ void RenewTreeOutputCUDAKernel_RegressionQuantile(
  const double* score,
  const label_t* label,
  const label_t* weight,
  double* residual_buffer,
  label_t* weight_by_leaf,
  double* weight_prefix_sum_buffer,
  const data_size_t* data_indices_in_leaf,
  const data_size_t* num_data_in_leaf,
  const data_size_t* data_start_in_leaf,
  data_size_t* data_indices_buffer,
  double* leaf_value,
  const double alpha) {
  const int leaf_index = static_cast<int>(blockIdx.x);
  const data_size_t data_start = data_start_in_leaf[leaf_index];
  const data_size_t num_data = num_data_in_leaf[leaf_index];
  data_size_t* data_indices_buffer_pointer = data_indices_buffer + data_start;
  const label_t* weight_by_leaf_pointer = weight_by_leaf + data_start;
  double* weight_prefix_sum_buffer_pointer = weight_prefix_sum_buffer + data_start;
  const double* residual_buffer_pointer = residual_buffer + data_start;
  for (data_size_t inner_data_index = data_start + static_cast<data_size_t>(threadIdx.x); inner_data_index < data_start + num_data; inner_data_index += static_cast<data_size_t>(blockDim.x)) {
    const data_size_t data_index = data_indices_in_leaf[inner_data_index];
    const label_t data_label = label[data_index];
    const double data_score = score[data_index];
    residual_buffer[inner_data_index] = static_cast<double>(data_label) - data_score;
    if (USE_WEIGHT) {
      weight_by_leaf[inner_data_index] = weight[data_index];
    }
  }
  __syncthreads();
  const double renew_leaf_value = PercentileDevice<double, data_size_t, label_t, double, false, USE_WEIGHT>(
    residual_buffer_pointer, weight_by_leaf_pointer, data_indices_buffer_pointer,
    weight_prefix_sum_buffer_pointer, alpha, num_data);
  if (threadIdx.x == 0) {
    leaf_value[leaf_index] = renew_leaf_value;
  }
}

void CUDARegressionQuantileloss::LaunchRenewTreeOutputCUDAKernel(
  const double* score, const data_size_t* data_indices_in_leaf, const data_size_t* num_data_in_leaf,
  const data_size_t* data_start_in_leaf, const int num_leaves, double* leaf_value) const {
  if (cuda_weights_ == nullptr) {
    RenewTreeOutputCUDAKernel_RegressionQuantile<false><<<num_leaves, GET_GRADIENTS_BLOCK_SIZE_REGRESSION / 2>>>(
      score,
      cuda_labels_,
      cuda_weights_,
      cuda_residual_buffer_.RawData(),
      cuda_weight_by_leaf_buffer_.RawData(),
      cuda_weights_prefix_sum_.RawData(),
      data_indices_in_leaf,
      num_data_in_leaf,
      data_start_in_leaf,
      cuda_data_indices_buffer_.RawData(),
      leaf_value,
      alpha_);
  } else {
    RenewTreeOutputCUDAKernel_RegressionQuantile<true><<<num_leaves, GET_GRADIENTS_BLOCK_SIZE_REGRESSION / 4>>>(
      score,
      cuda_labels_,
      cuda_weights_,
      cuda_residual_buffer_.RawData(),
      cuda_weight_by_leaf_buffer_.RawData(),
      cuda_weights_prefix_sum_.RawData(),
      data_indices_in_leaf,
      num_data_in_leaf,
      data_start_in_leaf,
      cuda_data_indices_buffer_.RawData(),
      leaf_value,
      alpha_);
  }
  SynchronizeCUDADevice(__FILE__, __LINE__);
}

template <bool USE_WEIGHT>
__global__ void GetGradientsKernel_RegressionQuantile(const double* cuda_scores, const label_t* cuda_labels,
  const label_t* cuda_weights, const data_size_t num_data, const double alpha,
  score_t* cuda_out_gradients, score_t* cuda_out_hessians) {
  const data_size_t data_index = static_cast<data_size_t>(blockDim.x * blockIdx.x + threadIdx.x);
  if (data_index < num_data) {
    if (!USE_WEIGHT) {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      if (diff >= 0.0f) {
        cuda_out_gradients[data_index] = (1.0f - alpha);
      } else {
        cuda_out_gradients[data_index] = -alpha;
      }
      cuda_out_hessians[data_index] = 1.0f;
    } else {
      const double diff = cuda_scores[data_index] - static_cast<double>(cuda_labels[data_index]);
      const score_t weight = static_cast<score_t>(cuda_weights[data_index]);
      if (diff >= 0.0f) {
        cuda_out_gradients[data_index] = (1.0f - alpha) * weight;
      } else {
        cuda_out_gradients[data_index] = -alpha * weight;
      }
      cuda_out_hessians[data_index] = weight;
    }
  }
}

void CUDARegressionQuantileloss::LaunchGetGradientsKernel(const double* score, score_t* gradients, score_t* hessians) const {
  const int num_blocks = (num_data_ + GET_GRADIENTS_BLOCK_SIZE_REGRESSION - 1) / GET_GRADIENTS_BLOCK_SIZE_REGRESSION;
  if (cuda_weights_ == nullptr) {
    GetGradientsKernel_RegressionQuantile<false><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, nullptr, num_data_, alpha_, gradients, hessians);
  } else {
    GetGradientsKernel_RegressionQuantile<true><<<num_blocks, GET_GRADIENTS_BLOCK_SIZE_REGRESSION>>>(score, cuda_labels_, cuda_weights_, num_data_, alpha_, gradients, hessians);
  }
}

}  // namespace LightGBM

#endif  // USE_CUDA
